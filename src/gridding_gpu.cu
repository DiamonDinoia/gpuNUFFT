#include "hip/hip_runtime.h"
#include "gridding_gpu.hpp"

#include "cuda_utils.hpp"

//Simple Test Kernel 
#define N 1000 //DIM ^3 
#define DIM 10
__global__ void kernel_call(int *a)
{
    int tx = threadIdx.x;
		int ty = threadIdx.y;
		int tz = threadIdx.z;

		int index = tx + DIM * (ty + tz * DIM);
		
		while (index < N)
		{
			a[index] = index;
			tz += blockDim.z;
			index = tx + DIM * (ty + tz * DIM);
		}
}

__constant__ GriddingInfo GI;
//extern __shared__ DType sdata_arr[];

#define N_THREADS_PER_SECTOR 5 //16x16
#define SECTOR_WIDTH 10

__global__ void griddingKernel( DType* data, 
							    DType* crds, 
							    DType* gdata,
							    DType* kernel, 
							    int* sectors, 
								  int* sector_centers,
								  DType* temp_gdata
								)
{
	__shared__ float sdata[2*SECTOR_WIDTH*SECTOR_WIDTH*SECTOR_WIDTH]; //ca. 8kB -> 2 Bl�cke je SM ???

	int sec= blockIdx.x;
	//TODO static or dynamic?
	//manually cast to correct type/pos
	//DType* sdata = (DType*)sdata_arr;
	for (int i=0; i<2*SECTOR_WIDTH*SECTOR_WIDTH*SECTOR_WIDTH;i++)
		sdata[i]=0.0f;

	if (sec < GI.sector_count)
	{
		int ind, max_x, max_y, max_z, imin, imax, jmin, jmax,kmin,kmax, k, i, j;

		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

			//Grid Points over threads
			int data_cnt;
			data_cnt = sectors[sec];
			
			while (data_cnt < sectors[sec+1])
			{
				__shared__ DType3 data_point; //datapoint shared in every thread
				data_point.x = crds[3*data_cnt];
				data_point.y = crds[3*data_cnt +1];
				data_point.z = crds[3*data_cnt +2];

				max_x = GI.sector_pad_width-1;
				max_y = GI.sector_pad_width-1;
				max_z = GI.sector_pad_width-1;

				// set the boundaries of final dataset for gridding this point
				ix = (data_point.x + 0.5f) * (GI.width) - center.x + GI.sector_offset;
				set_minmax(ix, &imin, &imax, max_x, GI.kernel_radius);
				jy = (data_point.y + 0.5f) * (GI.width) - center.y + GI.sector_offset;
				set_minmax(jy, &jmin, &jmax, max_y, GI.kernel_radius);
				kz = (data_point.z + 0.5f) * (GI.width) - center.z + GI.sector_offset;
				set_minmax(kz, &kmin, &kmax, max_z, GI.kernel_radius);

				// grid this point onto the neighboring cartesian points
				for (k=threadIdx.z;k<=kmax; k += blockDim.z)
				{
					__syncthreads();
					if (k<=kmax && k>=kmin)
					{
						kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.width)) - 0.5f;//(k - center_z) *width_inv;
						dz_sqr = kz - data_point.z;
						dz_sqr *= dz_sqr;
						if (dz_sqr < GI.radiusSquared)
						{
							j=threadIdx.y;
							if (j<=jmax && j>=jmin)
							{
								jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.width)) - 0.5f;   //(j - center_y) *width_inv;
								dy_sqr = jy - data_point.y;
								dy_sqr *= dy_sqr;
								if (dy_sqr < GI.radiusSquared)	
								{
									i=threadIdx.x;
								
									if (i<=imax && i>=imin)
									{
										ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.width)) - 0.5f;// (i - center_x) *width_inv;
										dx_sqr = ix - data_point.x;
										dx_sqr *= dx_sqr;
										if (dx_sqr < GI.radiusSquared)	
										{
											// get kernel value
											//Berechnung mit Separable Filters 
											val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
													kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
													kernel[(int) round(dx_sqr * GI.dist_multiplier)];
											ind = 2* getIndex(i,j,k,GI.sector_pad_width);
								
											// multiply data by current kernel val 
											// grid complex or scalar 
											sdata[ind]   += val * data[2*data_cnt];
											sdata[ind+1] += val * data[2*data_cnt+1];
										} // kernel bounds check x, spherical support 
									} // x 	 
								} // kernel bounds check y, spherical support 
							} // y 
						} //kernel bounds check z 
					} // z
				}//for loop over z entries
				__syncthreads();
				data_cnt++;
			} //grid points per sector
	
		int sector_ind_offset = sec * GI.sector_pad_width*GI.sector_pad_width*GI.sector_pad_width;
		for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
		{
			__syncthreads();
			int y=threadIdx.y;
			int x=threadIdx.x;
			
			int s_ind = 2* getIndex(x,y,z,GI.sector_pad_width) ;
			ind = 2*(sector_ind_offset + getIndex(x,y,z,GI.sector_pad_width));
						
			temp_gdata[ind] = sdata[s_ind];//Re
			temp_gdata[ind+1] = sdata[s_ind+1];//Im
		}
	}//sec < sector_count	
}

__global__ void composeOutput(DType* temp_gdata, DType* gdata, int* sector_centers)
{
	for (int sec = 0; sec < GI.sector_count; sec++)
	{
		int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];
		int sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.width);
		
		int sector_grid_offset = sec * GI.sector_pad_width*GI.sector_pad_width*GI.sector_pad_width;
			
		for (int z = 0; z < GI.sector_pad_width; z++)
			for (int y = 0; y < GI.sector_pad_width; y++)
			{
				for (int x = 0; x < GI.sector_pad_width; x++)
				{
					int s_ind = 2* (sector_grid_offset + getIndex(x,y,z,GI.sector_pad_width));
					int ind = 2*(sector_ind_offset + getIndex(x,y,z,GI.width));
					//TODO outlier richtig???
					if (isOutlier(x,y,z,center.x,center.y,center.z,GI.width,GI.sector_offset))
						continue;

					gdata[ind] += temp_gdata[s_ind];//Re
					gdata[ind+1] += temp_gdata[s_ind+1];//Im
				}
			}
	}
}

void runSimpleKernelCall()
{
	printf("starting gpu implementation\n");
	int test_a[N];
	int* test_ad;

	allocateAndCopyToDeviceMem<int>(&test_ad,test_a,N);

	dim3 grid_size = 1;
	dim3 thread_size(10,10,3);
	printf("dimensions %d,%d,%d \n",thread_size.x,thread_size.y,thread_size.z);
	kernel_call<<<grid_size,thread_size>>>(test_ad);
	
	copyFromDevice<int>(test_ad,test_a,N);

	printf("output: ");
	for (int i = 0; i < N; i++)
		 printf("%d ",test_a[i]);
	printf("\n");
	
	freeDeviceMem(test_ad);
}

GriddingInfo* initAndCopyGriddingInfo(int sector_count, 
							 int sector_width,
							 int kernel_width,
							 int kernel_count, 
							 int width)
{
	GriddingInfo* gi_host = (GriddingInfo*)malloc(sizeof(GriddingInfo));

	gi_host->sector_count = sector_count;
	gi_host->sector_width = sector_width;
	
	gi_host->kernel_width = kernel_width; 
	gi_host->kernel_count = kernel_count;
	gi_host->width = width;

	DType kernel_radius = static_cast<DType>(kernel_width) / 2.0f;
	DType radius = kernel_radius / static_cast<DType>(width);
	DType width_inv = 1.0f / width;
	DType radiusSquared = radius * radius;
	DType kernelRadius_invSqr = 1 / radiusSquared;
	DType dist_multiplier = (kernel_count - 1) * kernelRadius_invSqr;
	printf("radius rel. to grid width %f\n",radius);
	int sector_pad_width = 10;//sector_width + 2*(int)(floor(kernel_width / 2.0f));
	int sector_dim = sector_pad_width  * sector_pad_width  * sector_pad_width ;
	int sector_offset = (int)(floor(sector_pad_width / 2.0f));

	gi_host->kernel_radius = kernel_radius;
	gi_host->sector_pad_width = sector_pad_width;
	gi_host->sector_dim = sector_dim;
	gi_host->sector_offset = sector_offset;
	gi_host->radiusSquared = radiusSquared;
	gi_host->dist_multiplier = dist_multiplier;

	printf("sector offset = %d\n",sector_offset);
	
	gi_host->sector_pad_width = sector_pad_width;
	
	printf("copy Gridding Info to symbol memory...\n");
	hipMemcpyToSymbol(HIP_SYMBOL(GI), gi_host,sizeof(GriddingInfo));
	//free(gi_host);
	printf("...done!\n");
	return gi_host;
}

void gridding3D_gpu(DType* data, 
					int data_cnt,
					DType* crds, 
					DType* gdata,
					int gdata_cnt,
					DType* kernel,
					int kernel_cnt,
					int* sectors, 
					int sector_count, 
					int* sector_centers,
					int sector_width,
					int kernel_width, 
					int kernel_count, 
					int width)
{
	//runSimpleKernelCall();
		
	assert(sectors != NULL);
	
	//split and run sectors into blocks
	//and each data point to one thread inside this block 

	GriddingInfo* gi_host = initAndCopyGriddingInfo(sector_count,sector_width,kernel_width,kernel_count,width);
	
	DType* data_d, *crds_d, *gdata_d, *kernel_d, *temp_gdata_d;
	int* sector_centers_d, *sectors_d;

	printf("allocate and copy gdata of size %d...\n",gdata_cnt);
	allocateAndCopyToDeviceMem<DType>(&gdata_d,gdata,gdata_cnt);//Konvention!!!

	printf("allocate and copy data of size %d...\n",2*data_cnt);
	allocateAndCopyToDeviceMem<DType>(&data_d,data,2*data_cnt);

	int temp_grid_cnt = 2 * sector_count * gi_host->sector_dim;
	//TODO delete
	DType* temp_gdata = (DType*) calloc(temp_grid_cnt,sizeof(DType));

	printf("allocate temp grid data of size %d...\n",temp_grid_cnt);
	allocateAndCopyToDeviceMem<DType>(&temp_gdata_d,temp_gdata,temp_grid_cnt);
	
	printf("allocate and copy coords of size %d...\n",3*data_cnt);
	allocateAndCopyToDeviceMem<DType>(&crds_d,crds,3*data_cnt);
	
	printf("allocate and copy kernel of size %d...\n",kernel_cnt);
	allocateAndCopyToDeviceMem<DType>(&kernel_d,kernel,kernel_cnt);
	printf("allocate and copy sectors of size %d...\n",2*sector_count);
	allocateAndCopyToDeviceMem<int>(&sectors_d,sectors,2*sector_count);
	printf("allocate and copy sector_centers of size %d...\n",3*sector_count);
	allocateAndCopyToDeviceMem<int>(&sector_centers_d,sector_centers,3*sector_count);
	
	dim3 block_dim(SECTOR_WIDTH,SECTOR_WIDTH,2);

  griddingKernel<<<sector_count,block_dim>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d);

	//compose total output from local blocks 
	composeOutput<<<1,1>>>(temp_gdata_d,gdata_d,sector_centers_d);

	//TODO Inverse fft


	//TODO deapodization


	copyFromDevice<DType>(gdata_d,gdata,gdata_cnt);
	
	freeDeviceMem(data_d);
	freeDeviceMem(crds_d);
	freeDeviceMem(gdata_d);
	freeDeviceMem(kernel_d);
	freeDeviceMem(sectors_d);
	freeDeviceMem(sector_centers_d);
	freeDeviceMem(temp_gdata_d);
	free(gi_host);
}