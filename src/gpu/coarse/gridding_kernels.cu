#include "hip/hip_runtime.h"
#include "gridding_kernels.hpp"
#include "cuda_utils.cuh"

__global__ void convolutionKernel( DType* data, 
							    DType* crds, 
							    CufftType* gdata,
							    DType* kernel, 
							    int* sectors, 
								int* sector_centers,
								DType* temp_gdata
								)
{
	extern __shared__ DType sdata[];//externally managed shared memory

	int sec= blockIdx.x;
	//init shared memory
	for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
	{
			int y=threadIdx.y;
			int x=threadIdx.x;
			int s_ind = 2* getIndex(x,y,z,GI.sector_pad_width) ;
			sdata[s_ind] = 0.0f;//Re
			sdata[s_ind+1]=0.0f;//Im
	}
	__syncthreads();
	//start convolution
	if (sec < GI.sector_count)
	{
		int ind, k, i, j;
		__shared__ int max_x, max_y, max_z, imin, imax,jmin,jmax,kmin,kmax;

		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Points over threads
		int data_cnt;
		data_cnt = sectors[sec];
			
		while (data_cnt < sectors[sec+1])
		{
			__shared__ DType3 data_point; //datapoint shared in every thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];

			max_x = GI.sector_pad_width-1;
			max_y = GI.sector_pad_width-1;
			max_z = GI.sector_pad_width-1;

			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(kz, &kmin, &kmax, max_z, GI.kernel_radius);
				                
			// grid this point onto the neighboring cartesian points
			for (k=threadIdx.z;k<=kmax; k += blockDim.z)
			{
				if (k<=kmax && k>=kmin)
				{
					kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
					dz_sqr = kz - data_point.z;
					dz_sqr *= dz_sqr;
					if (dz_sqr < GI.radiusSquared)
					{
						j=threadIdx.y;
						if (j<=jmax && j>=jmin)
						{
							jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
							dy_sqr = jy - data_point.y;
							dy_sqr *= dy_sqr;
							if (dy_sqr < GI.radiusSquared)	
							{
								i=threadIdx.x;
								
								if (i<=imax && i>=imin)
								{
									ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
									dx_sqr = ix - data_point.x;
									dx_sqr *= dx_sqr;
									if (dx_sqr < GI.radiusSquared)	
									{
										//get kernel value
										//Calculate Separable Filters 
										val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
											  kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
											  kernel[(int) round(dx_sqr * GI.dist_multiplier)];
										ind = 2* getIndex(i,j,k,GI.sector_pad_width);
								
										// multiply data by current kernel val 
										// grid complex or scalar 
										sdata[ind]   += val * data[2*data_cnt];
										sdata[ind+1] += val * data[2*data_cnt+1];
									} // kernel bounds check x, spherical support 
								} // x 	 
							} // kernel bounds check y, spherical support 
						} // y 
					} //kernel bounds check z 
				} // z
			}//for loop over z entries
			__syncthreads();
			data_cnt++;
		} //grid points per sector
	
	  //write shared data to temporary output grid
		int sector_ind_offset = sec * GI.sector_dim;
		for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
		{
			int y=threadIdx.y;
			int x=threadIdx.x;
			
			int s_ind = 2* getIndex(x,y,z,GI.sector_pad_width) ;//index in shared grid
			ind = 2*sector_ind_offset + s_ind;//index in temp output grid
						
			temp_gdata[ind] = sdata[s_ind];//Re
			temp_gdata[ind+1] = sdata[s_ind+1];//Im
		}
	}//sec < sector_count	
}

__global__ void composeOutputKernel(DType* temp_gdata, CufftType* gdata, int* sector_centers)
{
	for (int sec = 0; sec < GI.sector_count; sec++)
	{
		__syncthreads();
		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];
		__shared__ int sector_ind_offset;
		sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		__shared__ int sector_grid_offset;
		sector_grid_offset = sec * GI.sector_dim;
		//write data from temp grid to overall output grid
		for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
		{
			int y=threadIdx.y;
			int x=threadIdx.x;
			int s_ind = 2* (sector_grid_offset + getIndex(x,y,z,GI.sector_pad_width));
			int ind = (sector_ind_offset + getIndex(x,y,z,GI.grid_width));
			if (isOutlier(x,y,z,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
				continue;
			gdata[ind].x += temp_gdata[s_ind];//Re
			gdata[ind].y += temp_gdata[s_ind+1];//Im
		}
	}
}


__global__ void deapodizationKernel(CufftType* gdata, DType beta, DType norm_val)
{
	int x=blockIdx.x;
	int y=blockIdx.y;
	int z=threadIdx.x;

	int ind = getIndex(x,y,z,GI.grid_width);
	
	DType deapo = calculateDeapodizationAt(x,y,z,GI.grid_width_offset,GI.grid_width_inv,GI.kernel_width,beta,norm_val);
	
	//check if deapodization value is valid number
	if (!isnan(deapo))// == deapo)
	{
		gdata[ind].x = gdata[ind].x / deapo;//Re
		gdata[ind].y = gdata[ind].y / deapo;//Im
	}
}

__global__ void fftShiftKernel(CufftType* gdata, int offset)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int z = threadIdx.x;

	//calculate "opposite" coord pair
	int x_opp = (x + offset) % GI.grid_width;
	int y_opp = (y + offset) % GI.grid_width;
	int z_opp = (z + offset) % GI.grid_width;

	//swap points
	CufftType temp = gdata[getIndex(x,y,z,GI.grid_width)];
	gdata[getIndex(x,y,z,GI.grid_width)] = gdata[getIndex(x_opp,y_opp,z_opp,GI.grid_width)];
	gdata[getIndex(x_opp,y_opp,z_opp,GI.grid_width)] = temp;

}


void performConvolution( DType* data_d, 
						 DType* crds_d, 
						 CufftType* gdata_d,
						 DType* kernel_d, 
						 int* sectors_d, 
						 int* sector_centers_d,
						 DType* temp_gdata_d,
						 dim3 grid_dim,
						 dim3 block_dim,
						 GriddingInfo* gi_host
						)
{
	long shared_mem_size = 2*gi_host->sector_dim*sizeof(DType);
	printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);
	convolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d);
}

void composeOutput(DType* temp_gdata_d, CufftType* gdata_d, int* sector_centers_d,dim3 grid_dim,dim3 block_dim)
{
	composeOutputKernel<<<grid_dim,block_dim>>>(temp_gdata_d,gdata_d,sector_centers_d);
}

//see BEATTY et al.: RAPID GRIDDING RECONSTRUCTION
//eq. (4) and (5)
void performDeapodization(CufftType* gdata,
						 dim3 grid_dim,
						 dim3 block_dim,
						 GriddingInfo* gi_host)
{
	DType beta = (DType)BETA(gi_host->kernel_width,gi_host->osr);

	//Calculate normalization value (should be at position 0 in interval [-N/2,N/2]) 
	DType norm_val = calculateDeapodizationValue(0,gi_host->grid_width_inv,gi_host->kernel_width,beta);
	norm_val = norm_val * norm_val * norm_val;

	deapodizationKernel<<<grid_dim,block_dim>>>(gdata,beta,norm_val);
}

void performFFTShift(CufftType* gdata_d,
					 FFTShiftDir shift_dir,
					 int width)
{
	dim3 grid_dim((int)ceil(width/(DType)2.0));
	dim3 block_dim(width,width);
	int offset= 0;

	if (shift_dir == FORWARD)
	{
		offset = (int)ceil((DType)(width / (DType)2.0));
	}
	else
	{
		offset = (int)floor((DType)(width / (DType)2.0));
	}
	fftShiftKernel<<<block_dim,grid_dim>>>(gdata_d,offset);
}