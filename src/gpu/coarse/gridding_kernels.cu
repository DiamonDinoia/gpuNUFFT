#include "hip/hip_runtime.h"
#ifndef GRIDDING_KERNELS_CU
#define GRIDDING_KERNELS_CU

#include "gridding_kernels.hpp"
#include "../std_gridding_kernels.cu"
#include "cuda_utils.cuh"
#include "precomp_utils.hpp"

// ----------------------------------------------------------------------------
// convolutionKernel: NUFFT^H kernel
//
// Performs the gridding step by convolution of sample points with 
// interpolation function and resampling onto grid. 
//
// parameters:
//  * data           : complex input sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : output grid data 
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * temp_gdata     : temporary grid data
//  * N              : number of threads
__global__ void convolutionKernel(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  DType2* temp_gdata,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory

  int sec;
  sec = blockIdx.x;
  //init shared memory
  for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
  {
    int y=threadIdx.y;
    int x=threadIdx.x;
    int s_ind = getIndex(x,y,z,GI.sector_pad_width) ;
    sdata[s_ind].x = 0.0f;//Re
    sdata[s_ind].y = 0.0f;//Im
  }
  __syncthreads();
  //start convolution
  while (sec < N)
  {
    int ind, k, i, j;
    __shared__ int max_dim, imin, imax,jmin,jmax,kmin,kmax;

    DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

    __shared__ IndType3 center;
    center.x = sector_centers[sec * 3];
    center.y = sector_centers[sec * 3 + 1];
    center.z = sector_centers[sec * 3 + 2];//+ GI.aniso_z_shift;

    //Grid Points over threads
    int data_cnt;
    data_cnt = sectors[sec];

    max_dim =  GI.sector_pad_max;		
    
    while (data_cnt < sectors[sec+1])
    {
      __shared__ DType3 data_point; //datapoint shared in every thread
      data_point.x = crds[data_cnt];
      data_point.y = crds[data_cnt +GI.data_count];
      data_point.z = crds[data_cnt +2*GI.data_count];
      // set the boundaries of final dataset for gridding this point
      ix = static_cast<DType>((data_point.x + 0.5) * (GI.gridDims.x) - center.x + GI.sector_offset);
      set_minmax(&ix, &imin, &imax, max_dim, GI.kernel_radius);
      jy = static_cast<DType>((data_point.y + 0.5) * (GI.gridDims.y) - center.y + GI.sector_offset);
      set_minmax(&jy, &jmin, &jmax, max_dim, GI.kernel_radius);
      // take resolution in x(y) direction to keep isotropic voxel size
      //kz = static_cast<DType>((data_point.z + 0.5 - GI.aniso_z_shift) * (GI.gridDims.x) - center.z + GI.sector_offset);
      kz = static_cast<DType>((data_point.z + 0.5) * (GI.gridDims.z) - center.z + GI.sector_offset);
      set_minmax(&kz, &kmin, &kmax, max_dim, GI.kernel_radius);
      
      // grid this point onto the neighboring cartesian points
      for (k=threadIdx.z;k<=kmax; k += blockDim.z)
      {
        if (k<=kmax && k>=kmin)
        {
          //kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.x)) - 0.5f + GI.aniso_z_shift;
          kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.z)) - 0.5f;
          // scale distance in z direction with x,y dimension
          dz_sqr = (kz - data_point.z)*GI.aniso_z_scale;
          dz_sqr *= dz_sqr;
          if (dz_sqr < GI.radiusSquared)
          {
            j=threadIdx.y;
            if (j<=jmax && j>=jmin)
            {
              jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;
              dy_sqr = jy - data_point.y;
              dy_sqr *= dy_sqr;
              if (dy_sqr < GI.radiusSquared)	
              {
                i=threadIdx.x;

                if (i<=imax && i>=imin)
                {
                  ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;
                  dx_sqr = ix - data_point.x;
                  dx_sqr *= dx_sqr;
                  if (dx_sqr < GI.radiusSquared)	
                  {
                    //get kernel value
                    //Calculate Separable Filters 
                    val = KERNEL[(int) round(dz_sqr * GI.dist_multiplier)] *
                      KERNEL[(int) round(dy_sqr * GI.dist_multiplier)] *
                      KERNEL[(int) round(dx_sqr * GI.dist_multiplier)];
                    ind = getIndex(i,j,k,GI.sector_pad_width);
                    
                    // multiply data by current kernel val 
                    // grid complex or scalar 
                    sdata[ind].x += val * data[data_cnt].x;
                    sdata[ind].y += val * data[data_cnt].y;
                    
                  }  // kernel bounds check x, spherical support 
                } // x 	 
              }// kernel bounds check y, spherical support 
            } // y 
          } //kernel bounds check z 
        } // z
      }//for loop over z entries
      __syncthreads();	
      data_cnt++;
    } //grid points per sector
    __syncthreads();	
    //write shared data to temporary output grid
    int sector_ind_offset = sec * GI.sector_dim;
    
    for (k=threadIdx.z;k<GI.sector_pad_width; k += blockDim.z)
    {
      i=threadIdx.x;
      j=threadIdx.y;

      int s_ind = getIndex(i,j,k,GI.sector_pad_width) ;//index in shared grid
      ind = sector_ind_offset + s_ind;//index in temp output grid

      temp_gdata[ind].x = sdata[s_ind].x;//Re
      temp_gdata[ind].y = sdata[s_ind].y;//Im
      __syncthreads();
      sdata[s_ind].x = (DType)0.0;
      sdata[s_ind].y = (DType)0.0;
      __syncthreads();
    }
    __syncthreads();
    sec = sec + gridDim.x;
  }//sec < sector_count
  
}

__global__ void balancedConvolutionKernel(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors,
  IndType2* sector_processing_order,
  IndType* sector_centers,
  DType2* temp_gdata,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory

  int sec_cnt = blockIdx.x;
  int sec;
  //init shared memory
  for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
  {
    int y=threadIdx.y;
    int x=threadIdx.x;
    int s_ind = getIndex(x,y,z,GI.sector_pad_width) ;
    sdata[s_ind].x = 0.0f;//Re
    sdata[s_ind].y = 0.0f;//Im
  }
  __syncthreads();
  //start convolution
  while (sec_cnt < N)
  {
    sec = sector_processing_order[sec_cnt].x;
    int ind, k, i, j;
    __shared__ int max_dim, imin, imax,jmin,jmax,kmin,kmax;

    DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

    __shared__ IndType3 center;
    center.x = sector_centers[sec * 3];
    center.y = sector_centers[sec * 3 + 1];
    center.z = sector_centers[sec * 3 + 2];

    //Grid Points over threads
    int data_cnt;
    data_cnt = sectors[sec]+sector_processing_order[sec_cnt].y;

    max_dim =  GI.sector_pad_max;		
    
    while (data_cnt < sectors[sec+1])
    {
      __shared__ DType3 data_point; //datapoint shared in every thread
      data_point.x = crds[data_cnt];
      data_point.y = crds[data_cnt +GI.data_count];
      data_point.z = crds[data_cnt +2*GI.data_count];
      // set the boundaries of final dataset for gridding this point
      ix = static_cast<DType>((data_point.x + 0.5) * (GI.gridDims.x) - center.x + GI.sector_offset);
      set_minmax(&ix, &imin, &imax, max_dim, GI.kernel_radius);
      jy = static_cast<DType>((data_point.y + 0.5) * (GI.gridDims.y) - center.y + GI.sector_offset);
      set_minmax(&jy, &jmin, &jmax, max_dim, GI.kernel_radius);
      // take resolution in x(y) direction to keep isotropic voxel size
      //kz = static_cast<DType>((data_point.z + 0.5 - GI.aniso_z_shift) * (GI.gridDims.x) - center.z + GI.sector_offset);
      kz = static_cast<DType>((data_point.z + 0.5) * (GI.gridDims.z) - center.z + GI.sector_offset);
      set_minmax(&kz, &kmin, &kmax, max_dim, GI.kernel_radius);
      
      // grid this point onto the neighboring cartesian points
      for (k=threadIdx.z;k<=kmax; k += blockDim.z)
      {
        if (k<=kmax && k>=kmin)
        {
          //kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.x)) - 0.5f + GI.aniso_z_shift;
          kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.z)) - 0.5f;
          // scale distance in z direction with x,y dimension
          dz_sqr = (kz - data_point.z)*GI.aniso_z_scale;
          dz_sqr *= dz_sqr;
          if (dz_sqr < GI.radiusSquared)
          {
            j=threadIdx.y;
            if (j<=jmax && j>=jmin)
            {
              jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;
              dy_sqr = jy - data_point.y;
              dy_sqr *= dy_sqr;
              if (dy_sqr < GI.radiusSquared)	
              {
                i=threadIdx.x;

                if (i<=imax && i>=imin)
                {
                  ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;
                  dx_sqr = ix - data_point.x;
                  dx_sqr *= dx_sqr;
                  if (dx_sqr < GI.radiusSquared)	
                  {
                    //get kernel value
                    //Calculate Separable Filters 
                    val = KERNEL[(int) round(dz_sqr * GI.dist_multiplier)] *
                      KERNEL[(int) round(dy_sqr * GI.dist_multiplier)] *
                      KERNEL[(int) round(dx_sqr * GI.dist_multiplier)];
                    ind = getIndex(i,j,k,GI.sector_pad_width);
                    
                    // multiply data by current kernel val 
                    // grid complex or scalar 
                    sdata[ind].x += val * data[data_cnt].x;
                    sdata[ind].y += val * data[data_cnt].y;
                    
                  }  // kernel bounds check x, spherical support 
                } // x 	 
              }// kernel bounds check y, spherical support 
            } // y 
          } //kernel bounds check z 
        } // z
      }//for loop over z entries
      __syncthreads();	
      data_cnt++;
    } //grid points per sector
    __syncthreads();	
    //write shared data to temporary output grid
    int sector_ind_offset = sec * GI.sector_dim;
    
    for (k=threadIdx.z;k<GI.sector_pad_width; k += blockDim.z)
    {
      i=threadIdx.x;
      j=threadIdx.y;

      int s_ind = getIndex(i,j,k,GI.sector_pad_width) ;//index in shared grid
      ind = sector_ind_offset + s_ind;//index in temp output grid

      temp_gdata[ind].x = sdata[s_ind].x;//Re
      temp_gdata[ind].y = sdata[s_ind].y;//Im
      __syncthreads();
      sdata[s_ind].x = (DType)0.0;
      sdata[s_ind].y = (DType)0.0;
      __syncthreads();
    }
    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  }//sec < sector_count
  
}

__global__ void convolutionKernel2D(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  DType2* temp_gdata,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory

  int sec;
  sec = blockIdx.x;
  //init shared memory
  int y=threadIdx.y;
  int x=threadIdx.x;
  int s_ind = getIndex2D(x,y,GI.sector_pad_width) ;
  sdata[s_ind].x = 0.0f;//Re
  sdata[s_ind].y = 0.0f;//Im
  __syncthreads();
  //start convolution
  while (sec < N)
  {
    int ind, i, j;
    __shared__ int max_dim, imin, imax,jmin,jmax;

    DType dx_sqr, dy_sqr, val, ix, jy;

    __shared__ IndType2 center;
    center.x = sector_centers[sec * 2];
    center.y = sector_centers[sec * 2 + 1];

    //Grid Points over threads
    int data_cnt;
    data_cnt = sectors[sec];

    max_dim =  GI.sector_pad_max;		
    while (data_cnt < sectors[sec+1])
    {
      __shared__ DType2 data_point; //datapoint shared in every thread
      data_point.x = crds[data_cnt];
      data_point.y = crds[data_cnt +GI.data_count];
      // set the boundaries of final dataset for gridding this point
      ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
      set_minmax(&ix, &imin, &imax, max_dim, GI.kernel_radius);
      jy = (data_point.y + 0.5f) * (GI.gridDims.y) - center.y + GI.sector_offset;
      set_minmax(&jy, &jmin, &jmax, max_dim, GI.kernel_radius);

      // grid this point onto the neighboring cartesian points
      j=threadIdx.y;
      if (j<=jmax && j>=jmin)
      {
        jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;   
        dy_sqr = jy - data_point.y;
        dy_sqr *= dy_sqr;
        if (dy_sqr < GI.radiusSquared)	
        {
          i=threadIdx.x;

          if (i<=imax && i>=imin)
          {
            ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;
            dx_sqr = ix - data_point.x;
            dx_sqr *= dx_sqr;
            if (dx_sqr < GI.radiusSquared)	
            {
              //get kernel value
              //Calculate Separable Filters 
              val = KERNEL[(int) round(dy_sqr * GI.dist_multiplier)] *
                KERNEL[(int) round(dx_sqr * GI.dist_multiplier)];
              ind = getIndex2D(i,j,GI.sector_pad_width);

              // multiply data by current kernel val 
              // grid complex or scalar 
              sdata[ind].x += val * data[data_cnt].x;
              sdata[ind].y += val * data[data_cnt].y;
            } // kernel bounds check x, spherical support 
          } // x 	 
        } // kernel bounds check y, spherical support 
      } // y 
      __syncthreads();	
      data_cnt++;
    } //grid points per sector
    __syncthreads();	

    //write shared data to temporary output grid
    int sector_ind_offset = sec * GI.sector_dim;

    i=threadIdx.x;
    j=threadIdx.y;

    int s_ind = getIndex2D(i,j,GI.sector_pad_width) ;//index in shared grid
    ind = sector_ind_offset + s_ind;//index in temp output grid

    temp_gdata[ind].x = sdata[s_ind].x;//Re
    temp_gdata[ind].y = sdata[s_ind].y;//Im

    __syncthreads();
    sdata[s_ind].x = (DType)0.0;
    sdata[s_ind].y = (DType)0.0;

    __syncthreads();
    sec = sec + gridDim.x;
  }//sec < sector_count
}

__global__ void balancedConvolutionKernel2D(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType2* sector_processing_order,
  IndType* sector_centers,
  DType2* temp_gdata,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory
  
  int sec_cnt = blockIdx.x;
  int sec;

  //init shared memory
  int y=threadIdx.y;
  int x=threadIdx.x;
  int s_ind = getIndex2D(x,y,GI.sector_pad_width) ;
  sdata[s_ind].x = 0.0f;//Re
  sdata[s_ind].y = 0.0f;//Im
  __syncthreads();
  //start convolution
  while (sec_cnt < N)
  {
    sec = sector_processing_order[sec_cnt].x;
    int ind, i, j;
    __shared__ int max_dim, imin, imax,jmin,jmax;

    DType dx_sqr, dy_sqr, val, ix, jy;

    __shared__ IndType2 center;
    center.x = sector_centers[sec * 2];
    center.y = sector_centers[sec * 2 + 1];

    //Grid Points over threads
    int data_cnt;
    data_cnt = sectors[sec]+sector_processing_order[sec_cnt].y;

    max_dim =  GI.sector_pad_max;		
    while (data_cnt < sectors[sec+1])
    {
      __shared__ DType2 data_point; //datapoint shared in every thread
      data_point.x = crds[data_cnt];
      data_point.y = crds[data_cnt +GI.data_count];
      // set the boundaries of final dataset for gridding this point
      ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
      set_minmax(&ix, &imin, &imax, max_dim, GI.kernel_radius);
      jy = (data_point.y + 0.5f) * (GI.gridDims.y) - center.y + GI.sector_offset;
      set_minmax(&jy, &jmin, &jmax, max_dim, GI.kernel_radius);

      // grid this point onto the neighboring cartesian points
      j=threadIdx.y;
      if (j<=jmax && j>=jmin)
      {
        jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;   
        dy_sqr = jy - data_point.y;
        dy_sqr *= dy_sqr;
        if (dy_sqr < GI.radiusSquared)	
        {
          i=threadIdx.x;

          if (i<=imax && i>=imin)
          {
            ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;
            dx_sqr = ix - data_point.x;
            dx_sqr *= dx_sqr;
            if (dx_sqr < GI.radiusSquared)	
            {
              //get kernel value
              //Calculate Separable Filters 
              val = KERNEL[(int) round(dy_sqr * GI.dist_multiplier)] *
                KERNEL[(int) round(dx_sqr * GI.dist_multiplier)];
              ind = getIndex2D(i,j,GI.sector_pad_width);

              // multiply data by current kernel val 
              // grid complex or scalar 
              sdata[ind].x += val * data[data_cnt].x;
              sdata[ind].y += val * data[data_cnt].y;
            } // kernel bounds check x, spherical support 
          } // x 	 
        } // kernel bounds check y, spherical support 
      } // y 
      __syncthreads();	
      data_cnt++;
    } //grid points per sector
    __syncthreads();	

    //write shared data to temporary output grid
    int sector_ind_offset = sec * GI.sector_dim;

    i=threadIdx.x;
    j=threadIdx.y;

    int s_ind = getIndex2D(i,j,GI.sector_pad_width) ;//index in shared grid
    ind = sector_ind_offset + s_ind;//index in temp output grid

    temp_gdata[ind].x = sdata[s_ind].x;//Re
    temp_gdata[ind].y = sdata[s_ind].y;//Im

    __syncthreads();
    sdata[s_ind].x = (DType)0.0;
    sdata[s_ind].y = (DType)0.0;

    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  }//sec < sector_count
}


__global__ void composeOutputKernel(DType2* temp_gdata, CufftType* gdata, IndType* sector_centers)
{
  for (int sec = 0; sec < GI.sector_count; sec++)
  {
    __syncthreads();
    __shared__ IndType3 center;
    center.x = sector_centers[sec * 3];
    center.y = sector_centers[sec * 3 + 1];
    center.z = sector_centers[sec * 3 + 2];
    __shared__ int sector_ind_offset;

    sector_ind_offset = computeXYZ2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.gridDims);

    __shared__ int sector_grid_offset;
    sector_grid_offset = sec * GI.sector_dim;
    //write data from temp grid to overall output grid
    for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
    {
      int x=threadIdx.x;
      int y=threadIdx.y;
      int s_ind = (sector_grid_offset + getIndex(x,y,z,GI.sector_pad_width));

      int ind;
      if (isOutlier(x,y,z,center.x,center.y,center.z,GI.gridDims,GI.sector_offset))
      {
        //calculate opposite index
        ind = computeXYZ2Lin(calculateOppositeIndex(x,center.x,GI.gridDims.x,GI.sector_offset),
        calculateOppositeIndex(y,center.y,GI.gridDims.y,GI.sector_offset),
        calculateOppositeIndex(z,center.z,GI.gridDims.z,GI.sector_offset),
        GI.gridDims);
      }
      else
        ind = (sector_ind_offset + computeXYZ2Lin(x,y,z,GI.gridDims));

      gdata[ind].x += temp_gdata[s_ind].x;//Re
      gdata[ind].y += temp_gdata[s_ind].y;//Im			
    }
  }
}


__global__ void composeOutputKernel2D(DType2* temp_gdata, CufftType* gdata, IndType* sector_centers)
{
  for (int sec = 0; sec < GI.sector_count; sec++)
  {
    __syncthreads();
    __shared__ IndType2 center;
    center.x = sector_centers[sec * 2];
    center.y = sector_centers[sec * 2 + 1];
    __shared__ int sector_ind_offset;
    sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,GI.gridDims);
    __shared__ int sector_grid_offset;
    sector_grid_offset = sec * GI.sector_dim;
    //write data from temp grid to overall output grid
    int x=threadIdx.x;
    int y=threadIdx.y;
    int s_ind = (sector_grid_offset + getIndex2D(x,y,GI.sector_pad_width));

    int ind;
    if (isOutlier2D(x,y,center.x,center.y,GI.gridDims,GI.sector_offset))
      //calculate opposite index
      ind = computeXY2Lin(calculateOppositeIndex(x,center.x,GI.gridDims.x,GI.sector_offset),
      calculateOppositeIndex(y,center.y,GI.gridDims.y,GI.sector_offset),
      GI.gridDims);
    else
      ind = (sector_ind_offset + getIndex2D(x,y,GI.gridDims.x));

    gdata[ind].x += temp_gdata[s_ind].x;//Re
    gdata[ind].y += temp_gdata[s_ind].y;//Im			
  }
}


//very slow way of composing the output, should only be used on compute capabilties lower than 2.0
void composeOutput(DType2* temp_gdata_d, CufftType* gdata_d, IndType* sector_centers_d, GriddingND::GriddingInfo* gi_host)
{
  dim3 grid_dim(1);
  dim3 block_dim(gi_host->sector_pad_width,gi_host->sector_pad_width,1);
  if (gi_host->is2Dprocessing)
    composeOutputKernel2D<<<grid_dim,block_dim>>>(temp_gdata_d,gdata_d,sector_centers_d);
  else
    composeOutputKernel<<<grid_dim,block_dim>>>(temp_gdata_d,gdata_d,sector_centers_d);
}

void performConvolution( DType2* data_d, 
  DType* crds_d, 
  CufftType* gdata_d,
  DType* kernel_d, 
  IndType* sectors_d, 
  IndType* sector_centers_d,
  GriddingND::GriddingInfo* gi_host
  )
{
  DType2* temp_gdata_d;
  int temp_grid_count = gi_host->sector_count * gi_host->sector_dim;
  if (DEBUG)
    printf("allocate temp grid data of size %d...\n",temp_grid_count);
  allocateDeviceMem<DType2>(&temp_gdata_d,temp_grid_count);

  long shared_mem_size = gi_host->sector_dim*sizeof(DType2);

  dim3 block_dim(gi_host->sector_pad_width,gi_host->sector_pad_width,1);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,(gi_host->sector_pad_width)*(gi_host->sector_pad_width)*(1)));
  if (DEBUG)
    printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);

  if (gi_host->is2Dprocessing)
    convolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,temp_gdata_d,gi_host->sector_count);
  else
    convolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,temp_gdata_d,gi_host->sector_count);

  if (DEBUG && (hipDeviceSynchronize() != hipSuccess))
    printf("error at adj thread synchronization 2: %s\n",hipGetErrorString(hipGetLastError()));
  //compose total output from local blocks 
  composeOutput(temp_gdata_d,gdata_d,sector_centers_d,gi_host);
  
  if (DEBUG && (hipDeviceSynchronize() != hipSuccess))
    printf("error at adj thread synchronization 3: %s\n",hipGetErrorString(hipGetLastError()));
  
  freeDeviceMem((void*)temp_gdata_d);
}


void performConvolution( DType2* data_d, 
  DType* crds_d, 
  CufftType* gdata_d,
  DType* kernel_d, 
  IndType* sectors_d,
  IndType2* sector_processing_order_d,
  IndType* sector_centers_d,
  GriddingND::GriddingInfo* gi_host
  )
{
  DType2* temp_gdata_d;
  int temp_grid_count = gi_host->sector_count * gi_host->sector_dim;
  if (DEBUG)
    printf("allocate temp grid data of size %d...\n",temp_grid_count);
  allocateDeviceMem<DType2>(&temp_gdata_d,temp_grid_count);

  long shared_mem_size = gi_host->sector_dim*sizeof(DType2);

  dim3 block_dim(gi_host->sector_pad_width,gi_host->sector_pad_width,1);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,(gi_host->sector_pad_width)*(gi_host->sector_pad_width)*(1)));
  if (DEBUG)
    printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);

  if (gi_host->is2Dprocessing)
    balancedConvolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_processing_order_d,sector_centers_d,temp_gdata_d,gi_host->sectorsToProcess);
  else
    balancedConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_processing_order_d,sector_centers_d,temp_gdata_d,gi_host->sectorsToProcess);

  if (DEBUG && (hipDeviceSynchronize() != hipSuccess))
    printf("error at adj thread synchronization 2: %s\n",hipGetErrorString(hipGetLastError()));
  //compose total output from local blocks 
  composeOutput(temp_gdata_d,gdata_d,sector_centers_d,gi_host);
  
  if (DEBUG && (hipDeviceSynchronize() != hipSuccess))
    printf("error at adj thread synchronization 3: %s\n",hipGetErrorString(hipGetLastError()));
  
  freeDeviceMem((void*)temp_gdata_d);
}

// ----------------------------------------------------------------------------
// forwardConvolutionKernel: NUFFT kernel
//
// Performs the inverse gridding step by convolution of grid points with 
// interpolation function and resampling onto trajectory. 
//
// parameters:
//  * data           : complex output sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : input grid data 
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * N              : number of threads
__global__ void forwardConvolutionKernel(CufftType* data, 
  DType*     crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  int N)
{
  extern __shared__ CufftType shared_out_data[];//externally managed shared memory

  __shared__ int sec;
  sec = blockIdx.x;

  //init shared memory
  shared_out_data[threadIdx.x].x = 0.0f;//Re
  shared_out_data[threadIdx.x].y = 0.0f;//Im
  __syncthreads();
  //start convolution
  while (sec < N)
  {
    int ind, imin, imax, jmin, jmax,kmin,kmax, k, i, j;
    DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

    __shared__ IndType3 center;
    center.x = sector_centers[sec * 3];
    center.y = sector_centers[sec * 3 + 1];
    center.z = sector_centers[sec * 3 + 2];

    //Grid Points over Threads
    int data_cnt = sectors[sec] + threadIdx.x;

    __shared__ int sector_ind_offset;
    sector_ind_offset = computeXYZ2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.gridDims);

    while (data_cnt < sectors[sec+1])
    {
      DType3 data_point; //datapoint per thread
      data_point.x = crds[data_cnt];
      data_point.y = crds[data_cnt + GI.data_count];
      data_point.z = crds[data_cnt + 2*GI.data_count];

      // set the boundaries of final dataset for gridding this point
      ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
      set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
      jy = (data_point.y + 0.5f) * (GI.gridDims.x) - center.y + GI.sector_offset;
      set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);
      kz = (data_point.z + 0.5f) * (GI.gridDims.z) - center.z + GI.sector_offset;
      set_minmax(&kz, &kmin, &kmax, GI.sector_pad_max, GI.kernel_radius);

      // convolve neighboring cartesian points to this data point
      k = kmin;
      while (k<=kmax && k>=kmin)
      {
        kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.z)) - 0.5f;//(k - center_z) *width_inv;
        dz_sqr = (kz - data_point.z)*GI.aniso_z_scale;
        dz_sqr *= dz_sqr;

        if (dz_sqr < GI.radiusSquared)
        {
          j=jmin;
          while (j<=jmax && j>=jmin)
          {
            jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;   //(j - center_y) *width_inv;
            dy_sqr = jy - data_point.y;
            dy_sqr *= dy_sqr;
            if (dy_sqr < GI.radiusSquared)	
            {
              i=imin;								
              while (i<=imax && i>=imin)
              {
                ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;// (i - center_x) *width_inv;
                dx_sqr = ix - data_point.x;
                dx_sqr *= dx_sqr;
                if (dx_sqr < GI.radiusSquared)	
                {
                  // get kernel value
                  // calc as separable filter
                  val = KERNEL[(int) round(dz_sqr * GI.dist_multiplier)] *
                    KERNEL[(int) round(dy_sqr * GI.dist_multiplier)] *
                    KERNEL[(int) round(dx_sqr * GI.dist_multiplier)];

                  // multiply data by current kernel val 
                  // grid complex or scalar 
                  if (isOutlier(i,j,k,center.x,center.y,center.z,GI.gridDims,GI.sector_offset))
                    //calculate opposite index
                    ind = computeXYZ2Lin(calculateOppositeIndex(i,center.x,GI.gridDims.x,GI.sector_offset),
                    calculateOppositeIndex(j,center.y,GI.gridDims.y,GI.sector_offset),
                    calculateOppositeIndex(k,center.z,GI.gridDims.z,GI.sector_offset),
                    GI.gridDims);
                  else
                    ind = (sector_ind_offset + computeXYZ2Lin(i,j,k,GI.gridDims));

                  shared_out_data[threadIdx.x].x += gdata[ind].x * val; 
                  shared_out_data[threadIdx.x].y += gdata[ind].y * val;									
                }// kernel bounds check x, spherical support 
                i++;
              } // x loop
            } // kernel bounds check y, spherical support  
            j++;
          } // y loop
        } //kernel bounds check z 
        k++;
      } // z loop
      data[data_cnt].x = shared_out_data[threadIdx.x].x;
      data[data_cnt].y = shared_out_data[threadIdx.x].y;

      data_cnt = data_cnt + blockDim.x;

      shared_out_data[threadIdx.x].x = (DType)0.0;//Re
      shared_out_data[threadIdx.x].y = (DType)0.0;//Im
    } //data points per sector
    __syncthreads();
    sec = sec + gridDim.x;
  } //sector check
}

__global__ void forwardConvolutionKernel2D(CufftType* data, 
  DType*     crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  int N)
{
  extern __shared__ CufftType shared_out_data[];//externally managed shared memory

  __shared__ int sec;
  sec = blockIdx.x;

  //init shared memory
  shared_out_data[threadIdx.x].x = 0.0f;//Re
  shared_out_data[threadIdx.x].y = 0.0f;//Im
  __syncthreads();
  //start convolution
  while (sec < N)
  {
    int ind, imin, imax, jmin, jmax, i, j;
    DType dx_sqr, dy_sqr, val, ix, jy;

    __shared__ IndType2 center;
    center.x = sector_centers[sec * 2];
    center.y = sector_centers[sec * 2 + 1];

    //Grid Points over Threads
    int data_cnt = sectors[sec] + threadIdx.x;

    __shared__ int sector_ind_offset;
    sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,GI.gridDims);

    while (data_cnt < sectors[sec+1])
    {
      DType2 data_point; //datapoint per thread
      data_point.x = crds[data_cnt];
      data_point.y = crds[data_cnt + GI.data_count];

      // set the boundaries of final dataset for gridding this point
      ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
      set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
      jy = (data_point.y + 0.5f) * (GI.gridDims.x) - center.y + GI.sector_offset;
      set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

      // convolve neighboring cartesian points to this data point
      j=jmin;
      while (j<=jmax && j>=jmin)
      {
        jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;   //(j - center_y) *width_inv;
        dy_sqr = jy - data_point.y;
        dy_sqr *= dy_sqr;
        if (dy_sqr < GI.radiusSquared)	
        {
          i=imin;								
          while (i<=imax && i>=imin)
          {
            ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;// (i - center_x) *width_inv;
            dx_sqr = ix - data_point.x;
            dx_sqr *= dx_sqr;
            if (dx_sqr < GI.radiusSquared)	
            {
              // get kernel value
              // calc as separable filter
              val = KERNEL[(int) round(dy_sqr * GI.dist_multiplier)] *
                KERNEL[(int) round(dx_sqr * GI.dist_multiplier)];

              // multiply data by current kernel val 
              // grid complex or scalar 
              if (isOutlier2D(i,j,center.x,center.y,GI.gridDims.x,GI.sector_offset))
                //calculate opposite index
                ind = getIndex2D(calculateOppositeIndex(i,center.x,GI.gridDims.x,GI.sector_offset),
                calculateOppositeIndex(j,center.y,GI.gridDims.y,GI.sector_offset),
                GI.gridDims.x);
              else
                ind = (sector_ind_offset + getIndex2D(i,j,GI.gridDims.x));

              shared_out_data[threadIdx.x].x += gdata[ind].x * val; 
              shared_out_data[threadIdx.x].y += gdata[ind].y * val;									
            }// kernel bounds check x, spherical support 
            i++;
          } // x loop
        } // kernel bounds check y, spherical support  
        j++;
      } // y loop
      data[data_cnt].x = shared_out_data[threadIdx.x].x;
      data[data_cnt].y = shared_out_data[threadIdx.x].y;

      data_cnt = data_cnt + blockDim.x;

      shared_out_data[threadIdx.x].x = (DType)0.0;//Re
      shared_out_data[threadIdx.x].y = (DType)0.0;//Im
    } //data points per sector
    __syncthreads();
    sec = sec + gridDim.x;
  } //sector check
}


void performForwardConvolution( CufftType*		data_d, 
  DType*			crds_d, 
  CufftType*		gdata_d,
  DType*			kernel_d, 
  IndType*		sectors_d, 
  IndType*		sector_centers_d,
  GriddingND::GriddingInfo*	gi_host
  )
{
  int thread_size = THREAD_BLOCK_SIZE;
  long shared_mem_size = thread_size * sizeof(CufftType);//empiric

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,thread_size));

  if (DEBUG)
    printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);
  if (gi_host->is2Dprocessing)
    forwardConvolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,gi_host->sector_count);
  else
    forwardConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,gi_host->sector_count);
}

#endif //GRIDDING_KERNELS_CU