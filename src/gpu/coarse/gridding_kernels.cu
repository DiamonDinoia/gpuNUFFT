#include "hip/hip_runtime.h"
#include "gridding_kernels.hpp"
#include "cuda_utils.cuh"

// convolve every data point on grid position -> controlled by threadIdx.x .y and .z 
// shared data holds grid values as software managed cache
//
//
__global__ void convolutionKernel( DType* data, 
							    DType* crds, 
							    CufftType* gdata,
							    DType* kernel, 
							    int* sectors, 
								int* sector_centers,
								DType* temp_gdata
								)
{
	extern __shared__ DType sdata[];//externally managed shared memory

	int sec= blockIdx.x;
	//init shared memory
	for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
	{
			int y=threadIdx.y;
			int x=threadIdx.x;
			int s_ind = 2* getIndex(x,y,z,GI.sector_pad_width) ;
			sdata[s_ind] = 0.0f;//Re
			sdata[s_ind+1]=0.0f;//Im
	}
	__syncthreads();
	//start convolution
	if (sec < GI.sector_count)
	{
		int ind, k, i, j;
		__shared__ int max_x, max_y, max_z, imin, imax,jmin,jmax,kmin,kmax;

		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Points over threads
		int data_cnt;
		data_cnt = sectors[sec];
			
		while (data_cnt < sectors[sec+1])
		{
			__shared__ DType3 data_point; //datapoint shared in every thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];

			max_x = GI.sector_pad_width-1;
			max_y = GI.sector_pad_width-1;
			max_z = GI.sector_pad_width-1;

			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(kz, &kmin, &kmax, max_z, GI.kernel_radius);
				                
			// grid this point onto the neighboring cartesian points
			for (k=threadIdx.z;k<=kmax; k += blockDim.z)
			{
				if (k<=kmax && k>=kmin)
				{
					kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
					dz_sqr = kz - data_point.z;
					dz_sqr *= dz_sqr;
					if (dz_sqr < GI.radiusSquared)
					{
						j=threadIdx.y;
						if (j<=jmax && j>=jmin)
						{
							jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
							dy_sqr = jy - data_point.y;
							dy_sqr *= dy_sqr;
							if (dy_sqr < GI.radiusSquared)	
							{
								i=threadIdx.x;
								
								if (i<=imax && i>=imin)
								{
									ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
									dx_sqr = ix - data_point.x;
									dx_sqr *= dx_sqr;
									if (dx_sqr < GI.radiusSquared)	
									{
										//get kernel value
										//Calculate Separable Filters 
										val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
											  kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
											  kernel[(int) round(dx_sqr * GI.dist_multiplier)];
										ind = 2* getIndex(i,j,k,GI.sector_pad_width);
								
										// multiply data by current kernel val 
										// grid complex or scalar 
										sdata[ind]   += val * data[2*data_cnt];
										sdata[ind+1] += val * data[2*data_cnt+1];
									} // kernel bounds check x, spherical support 
								} // x 	 
							} // kernel bounds check y, spherical support 
						} // y 
					} //kernel bounds check z 
				} // z
			}//for loop over z entries
			__syncthreads();
			data_cnt++;
		} //grid points per sector
	
	  //write shared data to temporary output grid
		int sector_ind_offset = sec * GI.sector_dim;
		for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
		{
			int y=threadIdx.y;
			int x=threadIdx.x;
			
			int s_ind = 2* getIndex(x,y,z,GI.sector_pad_width) ;//index in shared grid
			ind = 2*sector_ind_offset + s_ind;//index in temp output grid
						
			temp_gdata[ind] = sdata[s_ind];//Re
			temp_gdata[ind+1] = sdata[s_ind+1];//Im
		}
	}//sec < sector_count	
}

__global__ void composeOutputKernel(DType* temp_gdata, CufftType* gdata, int* sector_centers)
{
	for (int sec = 0; sec < GI.sector_count; sec++)
	{
		__syncthreads();
		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];
		__shared__ int sector_ind_offset;
		sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		__shared__ int sector_grid_offset;
		sector_grid_offset = sec * GI.sector_dim;
		//write data from temp grid to overall output grid
		for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
		{
			int y=threadIdx.y;
			int x=threadIdx.x;
			int s_ind = 2* (sector_grid_offset + getIndex(x,y,z,GI.sector_pad_width));
			int ind = (sector_ind_offset + getIndex(x,y,z,GI.grid_width));
			if (isOutlier(x,y,z,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
				continue;
			gdata[ind].x += temp_gdata[s_ind];//Re
			gdata[ind].y += temp_gdata[s_ind+1];//Im
		}
	}
}


__global__ void deapodizationKernel(CufftType* gdata, DType beta, DType norm_val)
{
	int x=blockIdx.x;
	int y=blockIdx.y;
	int z=threadIdx.x;

	int ind = getIndex(x,y,z,GI.im_width);
	
	DType deapo = calculateDeapodizationAt(x,y,z,GI.im_width_offset,GI.grid_width_inv,GI.kernel_width,beta,norm_val);
	
	//check if deapodization value is valid number
	if (!isnan(deapo))// == deapo)
	{
		gdata[ind].x = gdata[ind].x / deapo;//Re
		gdata[ind].y = gdata[ind].y / deapo;//Im
	}
}

__global__ void cropKernel(CufftType* gdata,CufftType* imdata, int offset)
{
	int x=blockIdx.x; //[0,N-1] N...im_width
	int y=blockIdx.y; //[0,N-1] N...im_width
	int z=threadIdx.x;//[0,N-1] N...im_width
	int grid_ind = getIndex(offset+x,offset+y,offset+z,GI.grid_width);
	int im_ind = getIndex(x,y,z,GI.im_width);

	imdata[im_ind] = gdata[grid_ind];
}

__global__ void fftShiftKernel(CufftType* gdata, int offset)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int z = threadIdx.x;

	//calculate "opposite" coord pair
	int x_opp = (x + offset) % GI.grid_width;
	int y_opp = (y + offset) % GI.grid_width;
	int z_opp = (z + offset) % GI.grid_width;

	//swap points
	CufftType temp = gdata[getIndex(x,y,z,GI.grid_width)];
	gdata[getIndex(x,y,z,GI.grid_width)] = gdata[getIndex(x_opp,y_opp,z_opp,GI.grid_width)];
	gdata[getIndex(x_opp,y_opp,z_opp,GI.grid_width)] = temp;

}


void performConvolution( DType* data_d, 
						 DType* crds_d, 
						 CufftType* gdata_d,
						 DType* kernel_d, 
						 int* sectors_d, 
						 int* sector_centers_d,
						 DType* temp_gdata_d,
						 GriddingInfo* gi_host
						)
{
	long shared_mem_size = 2*gi_host->sector_dim*sizeof(DType);

	dim3 block_dim(gi_host->sector_pad_width,gi_host->sector_pad_width,N_THREADS_PER_SECTOR);
	dim3 grid_dim(gi_host->sector_count);
	
	printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);
	convolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d);
}

//very slow way of composing the output 
void composeOutput(DType* temp_gdata_d, CufftType* gdata_d, int* sector_centers_d, GriddingInfo* gi_host)
{
	dim3 grid_dim(1);
	dim3 block_dim(gi_host->sector_pad_width,gi_host->sector_pad_width,N_THREADS_PER_SECTOR);
	
	composeOutputKernel<<<grid_dim,block_dim>>>(temp_gdata_d,gdata_d,sector_centers_d);
}

//see BEATTY et al.: RAPID GRIDDING RECONSTRUCTION
//eq. (4) and (5)
void performDeapodization(CufftType* imdata_d,
						  GriddingInfo* gi_host)
{
	DType beta = (DType)BETA(gi_host->kernel_width,gi_host->osr);

	dim3 grid_dim(gi_host->im_width,gi_host->im_width,1);	
	dim3 block_dim(gi_host->im_width);
	//Calculate normalization value (should be at position 0 in interval [-N/2,N/2]) 
	DType norm_val = calculateDeapodizationValue(0,gi_host->grid_width_inv,gi_host->kernel_width,beta);
	norm_val = norm_val * norm_val * norm_val;

	deapodizationKernel<<<grid_dim,block_dim>>>(imdata_d,beta,norm_val);
}


void performCrop(CufftType* gdata_d,
				 CufftType* imdata_d,
				 GriddingInfo* gi_host)
{
	/*crop data 
    ind_off = (a.params.im_width * (double(a.params.osr)-1) / 2) + 1;
    ind_start = ind_off;
    ind_end = ind_start + a.params.im_width -1;
    ress = m(ind_start:ind_end,ind_start:ind_end,ind_start:ind_end,:);*/
	int ind_off = (int)(gi_host->im_width * ((DType)gi_host->osr - 1.0f)/(DType)2);
	printf("start cropping image with offset %d\n",ind_off);

	dim3 grid_dim(gi_host->im_width,gi_host->im_width,1);
	dim3 block_dim(gi_host->im_width);
	cropKernel<<<grid_dim,block_dim>>>(gdata_d,imdata_d,ind_off);
}

void performFFTShift(CufftType* gdata_d,
					 FFTShiftDir shift_dir,
					 int width)
{
	dim3 grid_dim((int)ceil(width/(DType)2.0));
	dim3 block_dim(width,width);
	int offset= 0;

	if (shift_dir == FORWARD)
	{
		offset = (int)ceil((DType)(width / (DType)2.0));
	}
	else
	{
		offset = (int)floor((DType)(width / (DType)2.0));
	}
	fftShiftKernel<<<block_dim,grid_dim>>>(gdata_d,offset);
}


__global__ void forwardDeapodizationKernel(DType* imdata, DType beta, DType norm_val)
{
	int x=blockIdx.x;
	int y=blockIdx.y;
	int z=threadIdx.x;

	int ind = 2*getIndex(x,y,z,GI.im_width);
	
	DType deapo = calculateDeapodizationAt(x,y,z,GI.im_width_offset,GI.grid_width_inv,GI.kernel_width,beta,norm_val);
	
	//TODO reciporcal or not????
	//check if deapodization value is valid number
	if (!isnan(deapo))// == deapo)
	{
		imdata[ind] = imdata[ind] / deapo; // / deapo;//Re
		imdata[ind+1] = imdata[ind+1] / deapo ; /// deapo;//Im
	}
}

__global__ void paddingKernel(DType* imdata,CufftType* gdata, int offset)
{
	int x=blockIdx.x; //[0,N-1] N...im_width
	int y=blockIdx.y; //[0,N-1] N...im_width
	int z=threadIdx.x;//[0,N-1] N...im_width

	int grid_ind =  getIndex(offset + x,offset + y,offset +z,GI.grid_width);

	int im_ind = 2*getIndex(x,y,z,GI.im_width);

	gdata[grid_ind].x = imdata[im_ind];
	gdata[grid_ind].y = imdata[im_ind+1];
}

__global__ void forwardConvolutionKernel( CufftType* data, 
										  DType* crds, 
										  CufftType* gdata,
										  DType* kernel, 
										  int* sectors, 
										  int* sector_centers
											)
{
	extern __shared__ CufftType shared_out_data[];//externally managed shared memory
	//test
	CufftType out_data;


	int sec= blockIdx.x;
	//init shared memory
	//out_data[threadIdx.x].x = 0.0f;//Re
	//out_data[threadIdx.x].y = 0.0f;//Im

	out_data.x = 0.0f;//Re
	out_data.y = 0.0f;//Im

	__syncthreads();

	//start convolution
	if (sec < GI.sector_count)
	{
		//shared???
		int ind, max_x, max_y, max_z, imin, imax, jmin, jmax,kmin,kmax, k, i, j;
		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Points over Threads
		int data_cnt = sectors[sec] + threadIdx.x;
		//out_data[data_cnt].x = 0.0f;//Re
		//out_data[data_cnt].y = 0.0f;//Im
		out_data.x = 0.0f;//Re
		out_data.y = 0.0f;//Im
		//int sector_grid_offset = sec * GI.sector_dim;
		int sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		
		while (data_cnt < sectors[sec+1])
		{
			DType3 data_point; //datapoint per thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];

			max_x = GI.sector_pad_width-1;
			max_y = GI.sector_pad_width-1;
			max_z = GI.sector_pad_width-1;

			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(kz, &kmin, &kmax, max_z, GI.kernel_radius);

			// convolve neighboring cartesian points to this data point
			k = kmin;
			while (k<=kmax && k>=kmin)
			{
				kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
				dz_sqr = kz - data_point.z;
				dz_sqr *= dz_sqr;
				if (dz_sqr < GI.radiusSquared)
				{
					j=jmin;
					while (j<=jmax && j>=jmin)
					{
						jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
						dy_sqr = jy - data_point.y;
						dy_sqr *= dy_sqr;
						if (dy_sqr < GI.radiusSquared)	
						{
							i=imin;								
							while (i<=imax && i>=imin)
							{
								ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
								dx_sqr = ix - data_point.x;
								dx_sqr *= dx_sqr;
								if (dx_sqr < GI.radiusSquared)	
								{
									// get kernel value
									//Berechnung mit Separable Filters 
									val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
											kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
											kernel[(int) round(dx_sqr * GI.dist_multiplier)];
									
									ind = (sector_ind_offset + getIndex(i,j,k,GI.grid_width));

									// multiply data by current kernel val 
									// grid complex or scalar 
									if (isOutlier(i,j,k,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
									{
										i++;
										continue;
									}
				
									//out_data[data_cnt].x = 1.0f; //val * gdata[ind].x;
									//out_data[data_cnt].y = 1.0f; //val * gdata[ind].y;		
									out_data.x += gdata[ind].x * val; //+= /*val **/ gdata[ind].x;
									out_data.y -= gdata[ind].y * val; //+= /*val **/ gdata[ind].y;
									
								}// kernel bounds check x, spherical support 
								i++;
							} // x loop
						} // kernel bounds check y, spherical support  
						j++;
					} // y loop
				} //kernel bounds check z 
				k++;
			} // z loop
			//data[data_cnt] = out_data[data_cnt];
			data[data_cnt].x = out_data.x;// / sqrt((DType)GI.kernel_width*GI.kernel_width*GI.kernel_width);
			data[data_cnt].y = out_data.y;// / sqrt((DType)GI.kernel_width*GI.kernel_width*GI.kernel_width);
			
			data_cnt = data_cnt + blockDim.x;

			//out_data[data_cnt].x = (DType)0.0f;
			//out_data[data_cnt].y = (DType)0.0f;
			out_data.x = 0.0f;//Re
			out_data.y = 0.0f;//Im
			//data[data_cnt] = out_data[data_cnt];
			//data_cnt++;
		} //data points per sector
	} //sector check
}

//see BEATTY et al.: RAPID GRIDDING RECONSTRUCTION
//eq. (4) and (5)
void performForwardDeapodization(DType* imdata_d,
						  GriddingInfo* gi_host)
{
	DType beta = (DType)BETA(gi_host->kernel_width,gi_host->osr);

	dim3 grid_dim(gi_host->im_width,gi_host->im_width,1);	
	dim3 block_dim(gi_host->im_width);
	//Calculate normalization value (should be at position 0 in interval [-N/2,N/2]) 
	DType norm_val = calculateDeapodizationValue(0,gi_host->grid_width_inv,gi_host->kernel_width,beta);
	norm_val = norm_val * norm_val * norm_val;

	forwardDeapodizationKernel<<<grid_dim,block_dim>>>(imdata_d,beta,norm_val);
}

void performPadding(DType* imdata_d,
					CufftType* gdata_d,					
					GriddingInfo* gi_host)
{
	int ind_off = (int)(gi_host->im_width * ((DType)gi_host->osr -1.0f)/(DType)2.0f);

	printf("start cropping image with offset %d\n",ind_off);

	dim3 grid_dim(gi_host->im_width,gi_host->im_width,1);
	dim3 block_dim(gi_host->im_width);
	paddingKernel<<<grid_dim,block_dim>>>(imdata_d,gdata_d,ind_off);
}

void performForwardConvolution( CufftType*		data_d, 
								DType*			crds_d, 
								CufftType*		gdata_d,
								DType*			kernel_d, 
								int*			sectors_d, 
								int*			sector_centers_d,
								GriddingInfo*	gi_host
								)
{
	//TODO how to calculate shared_mem_size???, shared_mem_needed?
	long shared_mem_size = 128 * sizeof(CufftType);//empiric

	dim3 block_dim(128);
	dim3 grid_dim(gi_host->sector_count);
	
	printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);
	forwardConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d);
}