#include "gridding_kernels.cu"
#include "../std_gridding_kernels.cu"
#include "cuda_utils.hpp"
#include "gridding_gpu.hpp"
#include "cufft_config.hpp"
/** gridding3D_gpu
  * forward gridding from image to grid/k-space
  * TODO
  * NFFT
**/
void gridding3D_gpu(CufftType**	data,			//kspace data array 
					int			data_count,		//data count, samples per trajectory
					int			n_coils,		//number of coils 
					DType*		crds,			//
					DType*		imdata,			//
					int			imdata_count,	//			
					int			grid_width,		//
					DType*		kernel,			//
					int			kernel_count,	//
					int			kernel_width,	//
					int*		sectors,		//
					int			sector_count,	//
					int*		sector_centers,	//
					int			sector_width,	//
					int			im_width,		//
					DType		osr,			//
					const GriddingOutput gridding_out)
{
	showMemoryInfo();
	GriddingInfo* gi_host = initAndCopyGriddingInfo(sector_count,sector_width,kernel_width,kernel_count,grid_width,im_width,osr);

	//cuda mem allocation
	DType *imdata_d, *crds_d, *kernel_d;//, *temp_gdata_d;
	CufftType *gdata_d, *data_d;
	int* sector_centers_d, *sectors_d;
	
	if (DEBUG)
		printf("allocate and copy imdata of size %d...\n",2*imdata_count*n_coils);
	allocateAndCopyToDeviceMem<DType>(&imdata_d,imdata,2*imdata_count*n_coils);

	if (DEBUG)
		printf("allocate and copy gdata of size %d...\n",gi_host->grid_width_dim );

	allocateAndSetMem<CufftType>(&gdata_d, gi_host->grid_width_dim,0);

	if (DEBUG)
		printf("allocate and copy data of size %d...\n",data_count * n_coils);
	allocateDeviceMem<CufftType>(&data_d,data_count * n_coils);

	if (DEBUG)
		printf("allocate and copy coords of size %d...\n",3*data_count);
	allocateAndCopyToDeviceMem<DType>(&crds_d,crds,3*data_count);
	
	if (DEBUG)
		printf("allocate and copy kernel of size %d...\n",kernel_count);
	allocateAndCopyToDeviceMem<DType>(&kernel_d,kernel,kernel_count);
	if (DEBUG)
		printf("allocate and copy sectors of size %d...\n",sector_count+1);
	allocateAndCopyToDeviceMem<int>(&sectors_d,sectors,sector_count+1);
	if (DEBUG)
		printf("allocate and copy sector_centers of size %d...\n",3*sector_count);
	allocateAndCopyToDeviceMem<int>(&sector_centers_d,sector_centers,3*sector_count);
	if (DEBUG)
		printf("sector pad width: %d\n",gi_host->sector_pad_width);
	
	//Inverse fft plan and execution
	hipfftHandle fft_plan;
	if (DEBUG)
		printf("creating cufft plan with %d,%d,%d dimensions\n",gi_host->grid_width,gi_host->grid_width,gi_host->grid_width);
	hipfftResult res = hipfftPlan3d(&fft_plan, gi_host->grid_width,gi_host->grid_width,gi_host->grid_width, CufftTransformType) ;
	if (res != HIPFFT_SUCCESS) 
		printf("error on CUFFT Plan creation!!! %d\n",res);
	int err;

	//iterate over coils and compute result
	for (int coil_it = 0; coil_it < n_coils; coil_it++)
	{
		int data_coil_offset = coil_it * data_count;
		int im_coil_offset = 2 * coil_it * imdata_count;//gi_host->width_dim;
		//reset temp array
		//hipMemset(temp_gdata_d,0, sizeof(DType)*temp_grid_count);
		hipMemset(data_d,0, sizeof(CufftType)*data_count);
		hipMemset(gdata_d,0, sizeof(CufftType)*gi_host->grid_width_dim);
		
		// apodization Correction
		performForwardDeapodization(imdata_d + im_coil_offset,gi_host);
		
		// resize by oversampling factor and zero pad
		performPadding(imdata_d + im_coil_offset,gdata_d,gi_host);
		
		// shift image to get correct zero frequency position
		performFFTShift(gdata_d,FORWARD,gi_host->grid_width);
		
		// eventually free imdata_d
		// Forward FFT to kspace domain
		if (err=pt2CufftExec(fft_plan, gdata_d, gdata_d, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		{
			printf("cufft has failed with err %i \n",err);
		}
		
		performFFTShift(gdata_d,FORWARD,gi_host->grid_width);
		
		// convolution and resampling to non-standard trajectory
		performForwardConvolution(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,gi_host);

		//get result
		copyFromDevice<CufftType>(data_d, *data + data_coil_offset,data_count);
	}//iterate over coils
	hipDeviceSynchronize();
	// Destroy the cuFFT plan.
	freeTotalDeviceMemory(data_d,crds_d,gdata_d,imdata_d,kernel_d,sectors_d,sector_centers_d,NULL);//NULL as stop
	hipfftDestroy(fft_plan);
	free(gi_host);
}

/** gridding3D_gpu
  * adjoint gridding from k-space to grid
  * TODO
  * NFFT^H
**/
void gridding3D_gpu_adj(DType*		data,			//kspace data array 
						int			data_count,		//data count, samples per trajectory
						int			n_coils,		//number of coils 
						DType*		crds,			//
						CufftType**	imdata,			//
						int			imdata_count,	//			
						int			grid_width,		//
						DType*		kernel,			//
						int			kernel_count,	//
						int			kernel_width,	//
						int*		sectors,		//
						int			sector_count,	//
						int*		sector_centers,	//
						int			sector_width,	//
						int			im_width,		//
						DType		osr,			//
						const GriddingOutput gridding_out)
{
	assert(sectors != NULL);
	
	showMemoryInfo();

	//split and run sectors into blocks
	//and each data point to one thread inside this block 
	GriddingInfo* gi_host = initAndCopyGriddingInfo(sector_count,sector_width,kernel_width,kernel_count,grid_width,im_width,osr);
	
	DType* data_d, *crds_d, *kernel_d, *temp_gdata_d;
	CufftType *gdata_d, *imdata_d;
	int* sector_centers_d, *sectors_d;

	if (DEBUG)
		printf("allocate and copy imdata of size %d...\n",imdata_count);
	allocateAndCopyToDeviceMem<CufftType>(&imdata_d,*imdata,imdata_count);//Konvention!!!

	if (DEBUG)
		printf("allocate and copy gdata of size %d...\n",gi_host->grid_width_dim);
	allocateDeviceMem<CufftType>(&gdata_d,gi_host->grid_width_dim);

	if (DEBUG)
		printf("allocate and copy data of size %d...\n",2*data_count*n_coils);
	allocateAndCopyToDeviceMem<DType>(&data_d,data,2*data_count*n_coils);

	int temp_grid_count = 2 * sector_count * gi_host->sector_dim;
	if (DEBUG)
		printf("allocate temp grid data of size %d...\n",temp_grid_count);
	allocateDeviceMem<DType>(&temp_gdata_d,temp_grid_count);

	if (DEBUG)
		printf("allocate and copy coords of size %d...\n",3*data_count);
	allocateAndCopyToDeviceMem<DType>(&crds_d,crds,3*data_count);
	
	if (DEBUG)
		printf("allocate and copy kernel of size %d...\n",kernel_count);
	allocateAndCopyToDeviceMem<DType>(&kernel_d,kernel,kernel_count);
	if (DEBUG)
		printf("allocate and copy sectors of size %d...\n",sector_count+1);
	allocateAndCopyToDeviceMem<int>(&sectors_d,sectors,sector_count+1);
	if (DEBUG)
		printf("allocate and copy sector_centers of size %d...\n",3*sector_count);
	allocateAndCopyToDeviceMem<int>(&sector_centers_d,sector_centers,3*sector_count);
	if (DEBUG)
		printf("sector pad width: %d\n",gi_host->sector_pad_width);
	
	//Inverse fft plan and execution
	hipfftHandle fft_plan;
	if (DEBUG)
		printf("creating cufft plan with %d,%d,%d dimensions\n",gi_host->grid_width,gi_host->grid_width,gi_host->grid_width);
	hipfftResult res = hipfftPlan3d(&fft_plan, gi_host->grid_width,gi_host->grid_width,gi_host->grid_width, CufftTransformType) ;
	if (res != HIPFFT_SUCCESS) 
		printf("error on CUFFT Plan creation!!! %d\n",res);
	int err;

	//iterate over coils and compute result
	for (int coil_it = 0; coil_it < n_coils; coil_it++)
	{
		int data_coil_offset = 2 * coil_it * data_count;
		int im_coil_offset = coil_it * imdata_count;//gi_host->width_dim;
		//reset temp array
		hipMemset(temp_gdata_d,0, sizeof(DType)*temp_grid_count);
		hipMemset(gdata_d,0, sizeof(CufftType)*gi_host->grid_width_dim);
		
		performConvolution(data_d+data_coil_offset,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,gi_host);

		//compose total output from local blocks 
		composeOutput(temp_gdata_d,gdata_d,sector_centers_d,gi_host);
	
		if (gridding_out == CONVOLUTION)
		{
			if (DEBUG)
				printf("stopping output after CONVOLUTION step\n");
			//get output
			copyFromDevice<CufftType>(gdata_d,*imdata,gi_host->grid_width_dim);
			if (DEBUG)
				printf("test value at point zero: %f\n",(*imdata)[0].x);
			freeTotalDeviceMemory(data_d,crds_d,imdata_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop token

			free(gi_host);
			// Destroy the cuFFT plan.
			hipfftDestroy(fft_plan);
			return;
		}
		performFFTShift(gdata_d,INVERSE,gi_host->grid_width);
		//Inverse FFT
		if (err=pt2CufftExec(fft_plan, gdata_d, gdata_d, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("cufft has failed with err %i \n",err);
			printf("cuda error: %s\n", hipGetErrorString(hipGetLastError()));
		}
	
		if (gridding_out == FFT)
		{
			if (DEBUG)
				printf("stopping output after FFT step\n");
			//get output
			copyFromDevice<CufftType>(gdata_d,*imdata,gi_host->grid_width_dim);
			
			//free memory
			if (hipfftDestroy(fft_plan) != HIPFFT_SUCCESS)
				printf("error on destroying cufft plan\n");
			freeTotalDeviceMemory(data_d,crds_d,imdata_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop token
			free(gi_host);
			// Destroy the cuFFT plan.
			printf("last cuda error: %s\n", hipGetErrorString(hipGetLastError()));
			return;
		}

		performFFTShift(gdata_d,INVERSE,gi_host->grid_width);
		
		performCrop(gdata_d,imdata_d,gi_host);
		
		performDeapodization(imdata_d,gi_host);

		//get result
		copyFromDevice<CufftType>(imdata_d,*imdata+im_coil_offset,imdata_count);
	}//iterate over coils
	// Destroy the cuFFT plan.
	hipfftDestroy(fft_plan);
	hipDeviceSynchronize();
	freeTotalDeviceMemory(data_d,crds_d,gdata_d,imdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop
	free(gi_host);
}
