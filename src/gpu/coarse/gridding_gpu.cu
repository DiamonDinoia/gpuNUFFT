#include "gridding_kernels.cu"
#include "cuda_utils.hpp"

//TODO forward gridding from grid to k-space

//adjoint gridding from k-space to grid
void gridding3D_gpu(DType*		data,			//kspace data array 
					int			data_count,		//data count, samples per trajectory
					int			n_coils,		//number of coils 
					DType*		crds,			//
					CufftType*	imdata,			//
					int			imdata_count,	//			
					int			grid_width,		//
					DType*		kernel,			//
					int			kernel_count,	//
					int			kernel_width,	//
					int*		sectors,		//
					int			sector_count,	//
					int*		sector_centers,	//
					int			sector_width,	//
					int			im_width,		//
					DType		osr,			//
					const GriddingOutput gridding_out)
{
	assert(sectors != NULL);
	
	size_t free_mem = 0;
	size_t total_mem = 0;
	hipMemGetInfo(&free_mem, &total_mem);
	printf("memory usage, free: %lu total: %lu\n",free_mem,total_mem);
	
	//split and run sectors into blocks
	//and each data point to one thread inside this block 
	GriddingInfo* gi_host = initAndCopyGriddingInfo(sector_count,sector_width,kernel_width,kernel_count,grid_width,im_width,osr);
	
	DType* data_d, *crds_d, *kernel_d, *temp_gdata_d;
	CufftType *gdata_d, *imdata_d;
	int* sector_centers_d, *sectors_d;

	printf("allocate and copy imdata of size %d...\n",imdata_count);
	allocateAndCopyToDeviceMem<CufftType>(&imdata_d,imdata,imdata_count);//Konvention!!!

	printf("allocate and copy gdata of size %d...\n",gi_host->grid_width_dim);
	allocateDeviceMem<CufftType>(&gdata_d,gi_host->grid_width_dim);

	printf("allocate and copy data of size %d...\n",2*data_count*n_coils);
	allocateAndCopyToDeviceMem<DType>(&data_d,data,2*data_count*n_coils);

	int temp_grid_count = 2 * sector_count * gi_host->sector_dim;
	printf("allocate temp grid data of size %d...\n",temp_grid_count);
	allocateDeviceMem<DType>(&temp_gdata_d,temp_grid_count);

	printf("allocate and copy coords of size %d...\n",3*data_count);
	allocateAndCopyToDeviceMem<DType>(&crds_d,crds,3*data_count);
	
	printf("allocate and copy kernel of size %d...\n",kernel_count);
	allocateAndCopyToDeviceMem<DType>(&kernel_d,kernel,kernel_count);
	printf("allocate and copy sectors of size %d...\n",sector_count+1);
	allocateAndCopyToDeviceMem<int>(&sectors_d,sectors,sector_count+1);
	printf("allocate and copy sector_centers of size %d...\n",3*sector_count);
	allocateAndCopyToDeviceMem<int>(&sector_centers_d,sector_centers,3*sector_count);
	printf("sector pad width: %d\n",gi_host->sector_pad_width);
	
	//Inverse fft plan and execution
	hipfftHandle fft_plan;
	printf("creating cufft plan with %d,%d,%d dimensions\n",gi_host->grid_width,gi_host->grid_width,gi_host->grid_width);
	hipfftResult res = hipfftPlan3d(&fft_plan, gi_host->grid_width,gi_host->grid_width,gi_host->grid_width, HIPFFT_C2C) ;
	if (res != HIPFFT_SUCCESS) 
		printf("error on CUFFT Plan creation!!! %d\n",res);
	int err;

	//iterate over coils and compute result
	for (int coil_it = 0; coil_it < n_coils; coil_it++)
	{
		int data_coil_offset = 2 * coil_it * data_count;
		int im_coil_offset = coil_it * imdata_count;//gi_host->width_dim;
		//reset temp array
		hipMemset(temp_gdata_d,0, sizeof(DType)*temp_grid_count);
		hipMemset(gdata_d,0, sizeof(CufftType)*gi_host->grid_width_dim);
		
		performConvolution(data_d+data_coil_offset,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,gi_host);

		//compose total output from local blocks 
		composeOutput(temp_gdata_d,gdata_d,sector_centers_d,gi_host);
	
		if (gridding_out == CONVOLUTION)
		{
			printf("stopping output after CONVOLUTION step\n");
			//get output
			copyFromDevice<CufftType>(gdata_d,imdata,gi_host->grid_width_dim);
			printf("test value at point zero: %f\n",imdata[0].x);
			freeTotalDeviceMemory(data_d,crds_d,imdata_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop token

			free(gi_host);
			/* Destroy the cuFFT plan. */
			hipfftDestroy(fft_plan);
			return;
		}

		//Inverse FFT
		if (err=hipfftExecC2C(fft_plan, gdata_d, gdata_d, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("cufft has failed with err %i \n",err);
		}
	
		if (gridding_out == FFT)
		{
			printf("stopping output after FFT step\n");
			//get output
			copyFromDevice<CufftType>(gdata_d,imdata,gi_host->grid_width_dim);
			
			//free memory
			if (hipfftDestroy(fft_plan) != HIPFFT_SUCCESS)
				printf("error on destroying cufft plan\n");
			freeTotalDeviceMemory(data_d,crds_d,imdata_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop token
			free(gi_host);
			/* Destroy the cuFFT plan. */
			printf("last cuda error: %s\n", hipGetErrorString(hipGetLastError()));
			return;
		}

		performFFTShift(gdata_d,INVERSE,gi_host->grid_width);

		performCrop(gdata_d,imdata_d,gi_host);
		
		performDeapodization(imdata_d,gi_host);

		//get result
		copyFromDevice<CufftType>(imdata_d,imdata+im_coil_offset,imdata_count);
	}//iterate over coils

	// Destroy the cuFFT plan.
	hipfftDestroy(fft_plan);
	freeTotalDeviceMemory(data_d,crds_d,gdata_d,imdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop
	free(gi_host);
}
