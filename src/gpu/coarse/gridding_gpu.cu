#include "gridding_kernels.cu"
#include "cuda_utils.hpp"

//TODO forward gridding from grid to k-space

//adjoint gridding from k-space to grid
void gridding3D_gpu(DType*		data,			//kspace data array 
					int			data_count,		//data count, samples per trajectory
					int			n_coils,		//number of coils 
					DType*		crds,			//
					CufftType*	imdata,			//
					int			imdata_count,	//			
					int			grid_width,		//
					DType*		kernel,			//
					int			kernel_count,	//
					int			kernel_width,	//
					int*		sectors,		//
					int			sector_count,	//
					int*		sector_centers,	//
					int			sector_width,	//
					int			im_width,		//
					DType		osr,			//
					const GriddingOutput gridding_out)
{
	assert(sectors != NULL);
	/*hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);*/
	size_t free_mem = 0;
	size_t total_mem = 0;
	hipInit(0);
	hipDevice_t dev; 
	hipCtx_t ctx;
	hipDeviceGet(&dev,0);
	hipError_t cuRes;
    if ((cuRes = hipCtxCreate(&ctx,0,dev)) != hipSuccess)
    {
		printf("get device failed\n");
		printf("%d\n", cuRes);//hipGetErrorString(hipGetLastError()));

	}
	hipMemGetInfo(&free_mem, &total_mem);
	printf("memory usage, free: %lu total: %lu\n",free_mem,total_mem);
	

	//split and run sectors into blocks
	//and each data point to one thread inside this block 
	GriddingInfo* gi_host = initAndCopyGriddingInfo(sector_count,sector_width,kernel_width,kernel_count,grid_width,im_width,osr);
	
	DType* data_d, *crds_d, *kernel_d, *temp_gdata_d;
	CufftType *gdata_d, *imdata_d;
	int* sector_centers_d, *sectors_d;

	printf("allocate and copy imdata of size %d...\n",imdata_count);
	allocateAndCopyToDeviceMem<CufftType>(&imdata_d,imdata,imdata_count);//Konvention!!!

	printf("allocate and copy gdata of size %d...\n",gi_host->grid_width_dim);
	//allocateAndSetMem<CufftType>(&gdata_d,gi_host->grid_width_dim,0);//Konvention!!! set mem in loop
	allocateDeviceMem<CufftType>(&gdata_d,gi_host->grid_width_dim);

	printf("allocate and copy data of size %d...\n",2*data_count*n_coils);
	allocateAndCopyToDeviceMem<DType>(&data_d,data,2*data_count*n_coils);

	int temp_grid_count = 2 * sector_count * gi_host->sector_dim;
	printf("allocate temp grid data of size %d...\n",temp_grid_count);
	//allocateAndSetMem<DType>(&temp_gdata_d,temp_grid_count,0);
	allocateDeviceMem<DType>(&temp_gdata_d,temp_grid_count);

	printf("allocate and copy coords of size %d...\n",3*data_count);
	allocateAndCopyToDeviceMem<DType>(&crds_d,crds,3*data_count);
	
	printf("allocate and copy kernel of size %d...\n",kernel_count);
	allocateAndCopyToDeviceMem<DType>(&kernel_d,kernel,kernel_count);
	printf("allocate and copy sectors of size %d...\n",sector_count+1);
	allocateAndCopyToDeviceMem<int>(&sectors_d,sectors,sector_count+1);
	printf("allocate and copy sector_centers of size %d...\n",3*sector_count);
	allocateAndCopyToDeviceMem<int>(&sector_centers_d,sector_centers,3*sector_count);
	printf("sector pad width: %d\n",gi_host->sector_pad_width);
	
	//Inverse fft plan and execution
	hipfftHandle fft_plan;
	printf("creating cufft plan with %d,%d,%d dimensions\n",gi_host->grid_width,gi_host->grid_width,gi_host->grid_width);
	hipfftResult res = hipfftPlan3d(&fft_plan, gi_host->grid_width,gi_host->grid_width,gi_host->grid_width, HIPFFT_C2C) ;
	if (res != HIPFFT_SUCCESS) 
		printf("error on CUFFT Plan creation!!! %d\n",res);
	int err;

	//iterate over coils and compute result
	for (int coil_it = 0; coil_it < n_coils; coil_it++)
	{
		int data_coil_offset = 2 * coil_it * data_count;
		int im_coil_offset = coil_it * imdata_count;//gi_host->width_dim;
		//reset temp array
		hipMemset(temp_gdata_d,0, sizeof(DType)*temp_grid_count);
		hipMemset(gdata_d,0, sizeof(CufftType)*gi_host->grid_width_dim);
		//hipMemset(imdata_d,0, sizeof(CufftType)*imdata_count);
		
		performConvolution(data_d+data_coil_offset,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,gi_host);

		//compose total output from local blocks 
		composeOutput(temp_gdata_d,gdata_d,sector_centers_d,gi_host);
	
		if (gridding_out == CONVOLUTION)
		{
			printf("stopping output after CONVOLUTION step\n");
			//get output
			copyFromDevice<CufftType>(gdata_d,imdata,gi_host->grid_width_dim);
			printf("test value at point zero: %f\n",imdata[0].x);
			freeTotalDeviceMemory(data_d,crds_d,imdata_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop token

			free(gi_host);
			/* Destroy the cuFFT plan. */
			hipfftDestroy(fft_plan);
			hipCtxDestroy(ctx);
			return;
		}

		//Inverse FFT
		if (err=hipfftExecC2C(fft_plan, gdata_d, gdata_d, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
		{
			printf("cufft has failed with err %i \n",err);
		  //return;
		}
	
		if (gridding_out == FFT)
		{
			printf("stopping output after FFT step\n");
			//get output
			copyFromDevice<CufftType>(gdata_d,imdata,gi_host->grid_width_dim);
			//free memory
			freeTotalDeviceMemory(data_d,crds_d,imdata_d,gdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop token
			free(gi_host);
			/* Destroy the cuFFT plan. */
			hipfftDestroy(fft_plan);
			printf("last cuda error: %s\n", hipGetErrorString(hipGetLastError()));
			hipCtxDestroy(ctx);
			return;
		}

		performFFTShift(gdata_d,INVERSE,gi_host->grid_width);

		//TODO crop
		//if (grid_width != im_width)
		{
			performCrop(gdata_d,imdata_d,gi_host);
		}

		performDeapodization(imdata_d,gi_host);

		//get result
		copyFromDevice<CufftType>(imdata_d,imdata+im_coil_offset,imdata_count);
	}//iterate over coils

/*	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed,start,stop);
	printf("Time elapsed: %3.1fms\n",elapsed);
	hipEventDestroy(start);
	hipEventDestroy(stop);*/

	/* Destroy the cuFFT plan. */
	hipfftDestroy(fft_plan);
	freeTotalDeviceMemory(data_d,crds_d,gdata_d,imdata_d,kernel_d,sectors_d,sector_centers_d,temp_gdata_d,NULL);//NULL as stop
	free(gi_host);
	hipCtxDestroy(ctx);
}
