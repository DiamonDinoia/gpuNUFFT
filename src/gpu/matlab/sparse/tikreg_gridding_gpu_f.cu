#include "mex.h"
#include "matrix.h"
#include <math.h>
#include <complex>
#include <vector>

#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <hipblas.h>

#include <stdio.h>
#include <iostream>

#define MAX_BLOCK_SZ 512

#include "tikreg_gridding_kernels.cu"

/**
 * Adjoint Gridding using sparse Matrix
 * 
 * Extracted from FREIBURG Code 
 * 
*/
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
 	//check count of passed arguments
    if(nrhs != 11 ) 
	{
		printf("\nUsage:\n");
		return;
	} else if(nlhs>1) 
	{
		printf("Too many output arguments\n");
		return;
	}

    // fetching data from MATLAB
    int pcnt = 0;  
    const mxArray *Measurement;
    Measurement = prhs[pcnt++];//0...Daten       
    std::complex<float> *meas = ( std::complex<float> *) mxGetData(Measurement);

    const mxArray *ImageDim;
    ImageDim = prhs[pcnt++];//1...Image Dimensions
    float *image_dims = (float*) mxGetData(ImageDim);
	const mwSize *dims_imagedim = mxGetDimensions(ImageDim);
	mexPrintf("Test %d, %d\n",dims_imagedim[0],dims_imagedim[1]);
		
	const mxArray *Sn;
    Sn = prhs[pcnt++];//1...SN Map
    float *sn = ( float *) mxGetData(Sn);
    
	int numsens;
	const mxArray *NumSens;
	NumSens = prhs[pcnt++];//2...Anzahl Spulen
	float *num_sens = (float*) mxGetData(NumSens);
	numsens = (int) num_sens[0];
	mexPrintf("Number of Coils: %d\n",numsens);
	
    const int numdim =5;
    const int dims_sz[] = {2, (int)image_dims[0], (int)image_dims[1], (int)image_dims[2],numsens };//2x64x64x44
    int w = (int)dims_sz[1];//64
    int h = (int)dims_sz[2];//64
    int d = (int)dims_sz[3];//44
    int totsz = w*h*d;//64x64x44 = 180224
	
    const mxArray *Ipk_index;
    Ipk_index = prhs[pcnt++];//2...Index (Y)       
    const mwSize *dims_ipk = mxGetDimensions(Ipk_index);
    float *ipk_index = (float*) mxGetData(Ipk_index);

    const mxArray *Ipk_we;
    Ipk_we = prhs[pcnt++]; //3...Weight (Y)      
    std::complex<float> *ipk_we = (std::complex<float>*) mxGetData(Ipk_we);
  
    int numP = dims_ipk[0];//125
    int numK = dims_ipk[1];//11685
    
	int* the_index= new int[numP*numK];
    for(int i = 0; i < numP*numK; i++)
        the_index[i] = (int)(ipk_index[i]-1);

    const mxArray *Dims_pad;
    Dims_pad = prhs[pcnt++];//4...Dimension Kd - Bild 64x64x44
    float *dims_pad_d = (float*) mxGetData(Dims_pad);
    int w_pad = (int)dims_pad_d[0];
    int h_pad = (int)dims_pad_d[1];
    int d_pad = (int)dims_pad_d[2];
    int totsz_pad  = w_pad*h_pad*d_pad;
 
    const mxArray *BPidx;
    BPidx = prhs[pcnt++];  //5...Backprojection VXIdx  - Bildinhalt   
    int numVox= mxGetM(BPidx); //Anzahl Zeilen
    int * bpidx = (int*) mxGetData(BPidx);
    
    const mxArray *BPmidx;
    BPmidx = prhs[pcnt++]; //6...Backprojection MidX 
   
    const mxArray *BPweight;
    BPweight = prhs[pcnt++]; //7...Backprojection Weight  

    const mxArray *Params;
    Params = prhs[pcnt++]; //8... Parameter   
    float *params = (float*) mxGetData(Params);
    float lambda = params[1]; //Regularisierungsparam
    int device_num = (int) params[2]; //Device
    int VERBOSE = (int) params[4]; //Verbose-Mode
    
    if (VERBOSE == 1)  
        mexPrintf("gpuDevice: %i  lambda^2: %f\n",device_num,lambda);

    /**************** Init Cuda *****************/
    hipDevice_t dev; 
    
    if (hipCtxGetDevice(&dev) == hipSuccess)
    {
		//   hipCtx_t  pctx ;
		//   hipCtxPopCurrent(&pctx);	      
    }   
    mexPrintf("dev:%i\n",dev);
       
    // MALLOCs
    hipfftComplex *tmp1,*tmp2, *_r , *_meas, *_ipk_we;
	float* _sn;
	
    int *_the_index;
    hipfftHandle            plan;
    
	//output erzeugen
	plhs[0]             =  mxCreateNumericArray(numdim,(const mwSize*)dims_sz,mxGetClassID(Measurement),mxREAL);
     
    std::complex<float> *res = (std::complex<float> *) mxGetData(plhs[0]);
   	
	hipMalloc( (void **) &_meas,sizeof(hipfftComplex)*numsens*numK);
    hipMalloc( (void **) &tmp1,sizeof(hipfftComplex)*totsz_pad);
    hipMalloc( (void **) &tmp2,sizeof(hipfftComplex)*totsz_pad);

	hipMalloc( (void **) &_sn,sizeof(float)*totsz);
    hipMalloc( (void **) &_r,sizeof(hipfftComplex)*totsz*numsens);
	
    hipMalloc( (void **) &_ipk_we,sizeof(hipfftComplex)*numP*numK);
    hipMalloc( (void **) &_the_index,sizeof(int)*numP*numK);
	 
    hipDeviceSynchronize();
   
    hipMemset( tmp1,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset(  _r,0,sizeof(hipfftComplex)*totsz*numsens); 
     hipDeviceSynchronize();

     /************** copy data on device **********************/

     hipMemcpy( _meas, meas, sizeof(hipfftComplex)*numsens*numK, hipMemcpyHostToDevice);
     hipMemcpy( _ipk_we, ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyHostToDevice);
     hipMemcpy( _the_index, the_index, sizeof(int)*numP*numK, hipMemcpyHostToDevice);
   
     hipMemcpy( ipk_we, _ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyDeviceToHost);
     hipMemcpy( the_index, _the_index, sizeof(int)*numP*numK, hipMemcpyDeviceToHost);
     hipMemcpy( _sn, sn, sizeof(float)*totsz, hipMemcpyHostToDevice);

     hipDeviceSynchronize();
    
    if (VERBOSE == 1) 
        mexPrintf("numP: %i  numK: %i whd %i %i %i pad %i %i %i numsens: %i\n",numP,numK,w,h,d,w_pad,h_pad,d_pad,numsens);

    /************** copy bpidx on device **********************/
    int *_bpmidx;
    hipfftComplex *_bpweight;
    int *bpsize = (int*) malloc(sizeof(int)*numVox);
    int *bponset  = (int*) malloc(sizeof(int)*(numVox+1));
    int *_bpsize, *_bponset, *_bpidx;
    bponset[0] = 0;
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        bpsize[j] = mxGetM(Midx);
        bponset[j+1] = bponset[j] + bpsize[j];
    }
    
    int *tmp_bpmidx;
    hipfftComplex *tmp_bpweight;
    tmp_bpmidx = (int*) malloc(sizeof(int)*bponset[numVox]);
    tmp_bpweight = (hipfftComplex*) malloc(sizeof(hipfftComplex)*bponset[numVox]);
    if (tmp_bpmidx == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    if (tmp_bpweight == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        mxArray *Weight = mxGetCell(BPweight,j);
        int *midx = (int*)  mxGetData(Midx);
        hipfftComplex *bpwei = (hipfftComplex*) mxGetData(Weight);
        memcpy(tmp_bpmidx + bponset[j] , midx, sizeof(int)* bpsize[j]);
        memcpy(tmp_bpweight + bponset[j] , bpwei, sizeof(hipfftComplex)* bpsize[j]);    
    }
    
    hipMalloc( (void **) &_bpmidx,sizeof(int)* bponset[numVox]);
    hipMalloc( (void **) &_bpweight,sizeof(hipfftComplex)* bponset[numVox]);
      
    hipMemcpy(_bpmidx,tmp_bpmidx,sizeof(int)*bponset[numVox], hipMemcpyHostToDevice);
    hipMemcpy(_bpweight,tmp_bpweight,sizeof(hipfftComplex)*bponset[numVox], hipMemcpyHostToDevice);
 
    free(tmp_bpmidx);
    free(tmp_bpweight);

    hipMalloc( (void **) &_bpsize,sizeof(int)* numVox);   
    hipMalloc( (void **) &_bpidx,sizeof(int)* numVox);
    hipMalloc( (void **) &_bponset,sizeof(int)* numVox+1);    
    hipMemcpy(_bpsize,bpsize,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bpidx,bpidx,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bponset,bponset,sizeof(int)* numVox+1, hipMemcpyHostToDevice);
            
    if (VERBOSE == 1) {
        mexPrintf("num active Vox: %i\n",numVox);
		mexPrintf("creating cufft plan with %d %d %d dimensions\n",d_pad,h_pad,w_pad);
	}
    
	int err;	
	if (err=hipfftPlan3d(&plan, d_pad, h_pad, w_pad, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{
		mexPrintf("create cufft plan has failed with err %i \n",err);
		mexPrintf("%s\n", hipGetErrorString(hipGetLastError()));
		return;
	}
    // thread managements 
    int vx_block = 128;
    dim3 dimBlock_vx(vx_block,1);
    dim3 dimGrid_vx (numVox/vx_block + 1,1);
 
    dim3 dimBlock_dw(d,1);//d=64
    dim3 dimGrid_dw (w,h);//w=64,h=64

    dim3 dimBlock_sq(d,1);
    dim3 dimGrid_sq (w*h,1);

    // we need this because first fft fails
    int _res = hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD);
	
    if (VERBOSE == 1)
      mexPrintf("first fft call ret: %i\n", _res);
	
	hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset(_r,0, sizeof(hipfftComplex)*totsz*numsens);            
	
    // backproject measurement -- x=A'b
    for (int i = 0; i < numsens; i++)
    { 
		//reset data for next coil
        hipMemset(tmp1,0, sizeof(hipfftComplex)*totsz_pad);
		//convolution with Gridding Kernel and Sampling (precomputed)
        backprojVX<<<dimGrid_vx,dimBlock_vx>>>(_bpidx,_bponset,_bpweight,_bpmidx,_bpsize,_meas + i*numK, tmp1,numVox);
        //Inverse FFT
		if (err=hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
        {
            mexPrintf("cufft has failed with err %i \n",err);
			mexPrintf("%s\n", hipGetErrorString(hipGetLastError()));
            return;
        }
		//Multiplikation mit SN Matrix
        sn_mult<<<dimGrid_dw,dimBlock_dw>>>(tmp2,tmp2, _sn, w, h, d, w_pad, h_pad, d_pad);
		
		//add -> without sense 
		addcoiltores<<<dimGrid_dw,dimBlock_dw>>>(_r,tmp2, totsz*numsens,i*totsz);
     }
  
     hipMemcpy( res, _r, sizeof(hipfftComplex)*totsz*numsens,hipMemcpyDeviceToHost);    

    hipFree(tmp1);
    hipFree(tmp2);
	
	hipFree(_sn);
    hipFree(_r); 
    hipFree(_meas);
	
    hipFree(_ipk_we);
    hipFree(_the_index);
	
    hipFree(_bpmidx);
    hipFree(_bpweight);
    hipFree(_bpsize);
    hipFree(_bpidx);
    hipFree(_bponset);    
    
    hipfftDestroy(plan);
    free(bpsize);
    free(bponset);
 
	mexPrintf("%s\n", hipGetErrorString(hipGetLastError()));

     hipCtx_t  pctx ;
     hipCtxPopCurrent(&pctx);	
}













