#include "mex.h"
#include "matrix.h"
#include <math.h>
#include <complex>
#include <vector>

#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <hipblas.h>

#include <stdio.h>
#include <iostream>

#define MAX_BLOCK_SZ 512

#include "tikreg_gridding_kernels.cu"

/**
 * Forward Gridding using sparse Matrix
 * 
 * Extracted from FREIBURG Code 
 * 
*/
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
	//check count of passed arguments
    if(nrhs != 11 ) 
	{
		printf("\nUsage:\n");
		return;
	} else if(nlhs>1) 
	{
		printf("Too many output arguments\n");
		return;
	}

    // fetching data from MATLAB
    int pcnt = 0;  
    const mxArray *ImageData;
    ImageData = prhs[pcnt++];//0...Image Daten       
    std::complex<float> *img = ( std::complex<float> *) mxGetData(ImageData);

    const mxArray *ImageDim;
    ImageDim = prhs[pcnt++];//1...Image Dimensions
    float *image_dims = (float*) mxGetData(ImageDim);
	const mwSize *dims_imagedim = mxGetDimensions(ImageDim);
	mexPrintf("Test %d, %d\n",dims_imagedim[0],dims_imagedim[1]);
	
	const mxArray *Sn;
    Sn = prhs[pcnt++];//1...SN Map
    float *sn = ( float *) mxGetData(Sn);
    
	int numsens;
	const mxArray *NumSens;
	NumSens = prhs[pcnt++];//2...Anzahl Spulen
	float *num_sens = (float*) mxGetData(NumSens);
	numsens = (int) num_sens[0];
	mexPrintf("Number of Coils: %d\n",numsens);
	
    const int dims_sz[] = {2, (int)image_dims[0], (int)image_dims[1], (int)image_dims[2],numsens };//2x64x64x44
    int w = (int)dims_sz[1];//64
    int h = (int)dims_sz[2];//64
    int d = (int)dims_sz[3];//44
    int totsz = w*h*d;//64x64x44 = 180224
	
    const mxArray *Ipk_index;
    Ipk_index = prhs[pcnt++];//2...Index (Y)       
    const mwSize *dims_ipk = mxGetDimensions(Ipk_index);
    float *ipk_index = (float*) mxGetData(Ipk_index);

    const mxArray *Ipk_we;
    Ipk_we = prhs[pcnt++]; //3...Weight (Y)      
    std::complex<float> *ipk_we = (std::complex<float>*) mxGetData(Ipk_we);
  
    int numP = dims_ipk[0];//125
    int numK = dims_ipk[1];//11685
	
	//output dimensions
	const int numdim =3;
	const int dims_k[] = {2, numK, numsens};
    
	int* the_index= new int[numP*numK];
    for(int i = 0; i < numP*numK; i++)
        the_index[i] = (int)(ipk_index[i]-1);

    const mxArray *Dims_pad;
    Dims_pad = prhs[pcnt++];//4...Dimension Kd - Bild 64x64x44
    float *dims_pad_d = (float*) mxGetData(Dims_pad);
    int w_pad = (int)dims_pad_d[0];
    int h_pad = (int)dims_pad_d[1];
    int d_pad = (int)dims_pad_d[2];
    int totsz_pad  = w_pad*h_pad*d_pad;
 
    const mxArray *BPidx;
    BPidx = prhs[pcnt++];  //5...Backprojection VXIdx  - Bildinhalt   
    int numVox= mxGetM(BPidx); //Anzahl Zeilen
    int * bpidx = (int*) mxGetData(BPidx);
    
    //const mxArray *BPmidx;
    //BPmidx = prhs[pcnt++]; //6...Backprojection MidX 
    pcnt++;
    //const mxArray *BPweight;
    //BPweight = prhs[pcnt++]; //7...Backprojection Weight  
	pcnt++;
    const mxArray *Params;
    Params = prhs[pcnt++]; //8... Parameter   
    float *params = (float*) mxGetData(Params);
    float lambda = params[1]; //Regularisierungsparam
    int device_num = (int) params[2]; //Device
    int VERBOSE = (int) params[4]; //Verbose-Mode
    
    if (VERBOSE == 1)  
        mexPrintf("gpuDevice: %i  lambda^2: %f\n",device_num,lambda);

   /**************** Init Cuda *****************/
    hipDevice_t dev; 

    if (hipCtxGetDevice(&dev) == hipSuccess)
    {
		//   hipCtx_t  pctx ;
		//   hipCtxPopCurrent(&pctx);	      
    }   
    mexPrintf("dev:%i\n",dev);
       
    // MALLOCs    
    hipfftComplex *tmp1,*tmp2, *_r , *_img, *_ipk_we;
	float* _sn;
	
    int *_the_index;
    hipfftHandle            plan;
    
	//output erzeugen
	plhs[0]             =  mxCreateNumericArray(numdim,(const mwSize*)dims_k,mxGetClassID(ImageData),mxREAL);
     
    std::complex<float> *res = (std::complex<float> *) mxGetData(plhs[0]);
   	
	hipMalloc( (void **) &_img,sizeof(hipfftComplex)*totsz*numsens);
    hipMalloc( (void **) &tmp1,sizeof(hipfftComplex)*totsz_pad);
    hipMalloc( (void **) &tmp2,sizeof(hipfftComplex)*totsz_pad);

    hipMalloc( (void **) &_sn,sizeof(float)*totsz);
    hipMalloc( (void **) &_r,sizeof(hipfftComplex)*numK*numsens);
	
    hipMalloc( (void **) &_ipk_we,sizeof(hipfftComplex)*numP*numK);
    hipMalloc( (void **) &_the_index,sizeof(int)*numP*numK);

    hipDeviceSynchronize();
   
    hipMemset( tmp1,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset( _img,0,sizeof(hipfftComplex)*totsz*numsens);
	 
    hipDeviceSynchronize();
 
     /************** copy data on device **********************/
	 hipMemcpy( _img, img, sizeof(hipfftComplex)*numsens*totsz, hipMemcpyHostToDevice);
     hipMemcpy( _ipk_we, ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyHostToDevice);
     hipMemcpy( _the_index, the_index, sizeof(int)*numP*numK, hipMemcpyHostToDevice);
	 hipMemcpy( _sn, sn, sizeof(float)*totsz, hipMemcpyHostToDevice);
     
     hipMemcpy( ipk_we, _ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyDeviceToHost);
     hipMemcpy( the_index, _the_index, sizeof(int)*numP*numK, hipMemcpyDeviceToHost);
 
     hipDeviceSynchronize();
    
    if (VERBOSE == 1) 
        mexPrintf("numP: %i  numK: %i whd %i %i %i pad %i %i %i numsens: %i\n",numP,numK,w,h,d,w_pad,h_pad,d_pad,numsens);
          
    if (VERBOSE == 1) {
        mexPrintf("num active Vox: %i\n",numVox);    
    }
    
	int err;
	if (err=hipfftPlan3d(&plan, d_pad, h_pad, w_pad, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{
		mexPrintf("create cufft plan has failed with err %i \n",err);
		mexPrintf("%s\n", hipGetErrorString(hipGetLastError()));
		return;
	}
    // thread managements 
    int vx_block = 128;
    dim3 dimBlock_vx(vx_block,1);
    dim3 dimGrid_vx (numVox/vx_block + 1,1);
 
    dim3 dimBlock_dw(d,1);//d=64
    dim3 dimGrid_dw (w,h);//w=64,h=64

    dim3 dimBlock_sq(d,1);
    dim3 dimGrid_sq (w*h,1);
    
	// for sensing 
    int sens_block = 256;
    dim3 dimBlock_se(sens_block,1);
    dim3 dimGrid_se (numK/sens_block + 1,1);

    // we need this because first fft fails
    int _res = hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD);
	
    if (VERBOSE == 1)
      mexPrintf("first fft call ret: %i\n", _res);
	
	hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset(_r,0, sizeof(hipfftComplex)*numK*numsens);            
	
	if (VERBOSE == 1)
		mexPrintf("start forward gridding...\n");
    // do sens -- b=A x
    for (int i = 0; i < numsens; i++)
    { 
		//reset data for next coil
        hipMemset(tmp1,0, sizeof(hipfftComplex)*totsz_pad);
		
		//Multiplikation mit SN Matrix
        sn_mult<<<dimGrid_dw,dimBlock_dw>>>(tmp1,_img, _sn, w, h, d, w_pad, h_pad, d_pad);     
		
		//FT in k-space
        if (err=hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
        {
			mexPrintf("cufft has failed with err %i \n",err);
			mexPrintf("%s\n", hipGetErrorString(hipGetLastError()));
            return;
        }

        hipMemset(_r,0, sizeof(hipfftComplex)*numK*numsens);
        dosens<<<dimGrid_se,dimBlock_se>>>(_r,tmp2,_ipk_we,_the_index,numP,numK);
     }
  
    hipMemcpy( res, _r, sizeof(hipfftComplex)*numK*numsens,hipMemcpyDeviceToHost);    

    hipFree(tmp1);
    hipFree(tmp2);
	
    hipFree(_r); 
    hipFree(_img);
	hipFree(_sn);
	
    hipFree(_ipk_we);
    hipFree(_the_index);
	    
    hipfftDestroy(plan);
    
    hipCtx_t  pctx ;
    hipCtxPopCurrent(&pctx);	
}













