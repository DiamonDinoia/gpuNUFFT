#include "mex.h"
#include "matrix.h"
#include <math.h>
#include <complex>
#include <vector>

#ifdef __unix__ 
# include <unistd.h>
#elif defined _WIN32 
# include <windows.h>
#endif

#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <hipblas.h>


#include <stdio.h>
#include <string>
#include <iostream>


#include <string.h>

#ifdef __unix__ 
	#include <sys/time.h>
#elif defined _WIN32 
	#include <time.h>
#endif

#define GET_TIME(now) { \
   struct timeval t; \
   gettimeofday(&t, NULL); \
   now = t.tv_sec + t.tv_usec/1000000.0; \
}

#define MAX_BLOCK_SZ 512

#include "tikreg_gridding_kernels.cu"

#ifdef _WIN32 

	#if defined(_MSC_VER) || defined(_MSC_EXTENSIONS)
	  #define DELTA_EPOCH_IN_MICROSECS  11644473600000000Ui64
	#else
	  #define DELTA_EPOCH_IN_MICROSECS  11644473600000000ULL
	#endif

	struct timezone
	{
	  int  tz_minuteswest; /* minutes W of Greenwich */
	  int  tz_dsttime;     /* type of dst correction */
	};
 
	// Definition of a gettimeofday function
 
	int gettimeofday(struct timeval *tv, struct timezone *tz)
	{
	// Define a structure to receive the current Windows filetime
	  FILETIME ft;
 
	// Initialize the present time to 0 and the timezone to UTC
	  unsigned __int64 tmpres = 0;
	  static int tzflag = 0;
 
	  if (NULL != tv)
	  {
		GetSystemTimeAsFileTime(&ft);
 
	// The GetSystemTimeAsFileTime returns the number of 100 nanosecond 
	// intervals since Jan 1, 1601 in a structure. Copy the high bits to 
	// the 64 bit tmpres, shift it left by 32 then or in the low 32 bits.
		tmpres |= ft.dwHighDateTime;
		tmpres <<= 32;
		tmpres |= ft.dwLowDateTime;
 
	// Convert to microseconds by dividing by 10
		tmpres /= 10;
 
	// The Unix epoch starts on Jan 1 1970.  Need to subtract the difference 
	// in seconds from Jan 1 1601.
		tmpres -= DELTA_EPOCH_IN_MICROSECS;
 
	// Finally change microseconds to seconds and place in the seconds value. 
	// The modulus picks up the microseconds.
		tv->tv_sec = (long)(tmpres / 1000000UL);
		tv->tv_usec = (long)(tmpres % 1000000UL);
	  }
 
	  if (NULL != tz)
	  {
		if (!tzflag)
		{
		  _tzset();
		  tzflag++;
		}
  
	// Adjust for the timezone west of Greenwich
		  tz->tz_minuteswest = _timezone / 60;
		tz->tz_dsttime = _daylight;
	  }
 
	  return 0;
	}
#endif

/**
 * Forward Gridding using sparse Matrix
 * 
 * Extracted from FREIBURG Code 
 * 
*/
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
 	
    if(nrhs != 11 ) 
	{
		printf("\nUsage:\n");
		return;
	} else if(nlhs>1) 
	{
		printf("Too many output arguments\n");
		return;
	}

    //////////////////////////////////// fetching data from MATLAB
    int pcnt = 0;  
    const mxArray *ImageData;
    ImageData = prhs[pcnt++];//0...Image Daten       
    std::complex<float> *img = ( std::complex<float> *) mxGetData(ImageData);

    const mxArray *ImageDim;
    ImageDim = prhs[pcnt++];//1...Image Dimensions
    float *image_dims = (float*) mxGetData(ImageDim);
	const mwSize *dims_imagedim = mxGetDimensions(ImageDim);
	mexPrintf("Test %d, %d\n",dims_imagedim[0],dims_imagedim[1]);
	
	const mxArray *Sn;
    Sn = prhs[pcnt++];//1...SN Map
    float *sn = ( float *) mxGetData(Sn);
    
	int numsens;
	const mxArray *NumSens;
	NumSens = prhs[pcnt++];//2...Anzahl Spulen
	float *num_sens = (float*) mxGetData(NumSens);
	numsens = (int) num_sens[0];
	mexPrintf("Number of Coils: %d\n",numsens);
	
    const int dims_sz[] = {2, image_dims[0], image_dims[1], image_dims[2],numsens };//2x64x64x44
    int w = (int)dims_sz[1];//64
    int h = (int)dims_sz[2];//64
    int d = (int)dims_sz[3];//44
    int totsz = w*h*d;//64x64x44 = 180224
	
    const mxArray *Ipk_index;
    Ipk_index = prhs[pcnt++];//2...Index (Y)       
    const mwSize *dims_ipk = mxGetDimensions(Ipk_index);
    float *ipk_index = (float*) mxGetData(Ipk_index);

    const mxArray *Ipk_we;
    Ipk_we = prhs[pcnt++]; //3...Weight (Y)      
    std::complex<float> *ipk_we = (std::complex<float>*) mxGetData(Ipk_we);
  
    int numP = dims_ipk[0];//125
    int numK = dims_ipk[1];//11685
	
	//output dimensions
	const int numdim =3;
	const int dims_k[] = {2, numK, numsens};
    
	int* the_index= new int[numP*numK];
    for(int i = 0; i < numP*numK; i++)
        the_index[i] = (int)(ipk_index[i]-1);

    const mxArray *Dims_pad;
    Dims_pad = prhs[pcnt++];//4...Dimension Kd - Bild 64x64x44
    float *dims_pad_d = (float*) mxGetData(Dims_pad);
    int w_pad = (int)dims_pad_d[0];
    int h_pad = (int)dims_pad_d[1];
    int d_pad = (int)dims_pad_d[2];
    int totsz_pad  = w_pad*h_pad*d_pad;
 
    const mxArray *BPidx;
    BPidx = prhs[pcnt++];  //5...Backprojection VXIdx  - Bildinhalt   
    int numVox= mxGetM(BPidx); //Anzahl Zeilen
    int * bpidx = (int*) mxGetData(BPidx);
    
    const mxArray *BPmidx;
    BPmidx = prhs[pcnt++]; //6...Backprojection MidX 
   
    const mxArray *BPweight;
    BPweight = prhs[pcnt++]; //7...Backprojection Weight  

    const mxArray *Params;
    Params = prhs[pcnt++]; //8... Parameter   
    float *params = (float*) mxGetData(Params);
    float lambda = params[1]; //Regularisierungsparam
    int device_num = (int) params[2]; //Device
    int VERBOSE = (int) params[4]; //Verbose-Mode
    
    if (VERBOSE == 1)  
        mexPrintf("gpuDevice: %i  lambda^2: %f\n",device_num,lambda);

   /**************** Init Cuda *****************/
    
    hipError_t rv; 
    hipDevice_t dev; 
    
    if (hipCtxGetDevice(&dev) == hipSuccess)
    {
    //   hipCtx_t  pctx ;
    //   hipCtxPopCurrent(&pctx);	      
    }   
    
    mexPrintf("dev:%i\n",dev);
       
    /////////////////////////////////////// MALLOCs
    
    double start,finish;
     
    GET_TIME(start);
    
    hipfftComplex *tmp1,*tmp2, *_r , *_img, *_ipk_we;
	float* _sn;
	
    int *_the_index;
    hipfftHandle            plan;
    
	//output erzeugen
	plhs[0]             =  mxCreateNumericArray(numdim,(const mwSize*)dims_k,mxGetClassID(ImageData),mxREAL);
     
    std::complex<float> *res = (std::complex<float> *) mxGetData(plhs[0]);
   	
	hipMalloc( (void **) &_img,sizeof(hipfftComplex)*totsz*numsens);
    hipMalloc( (void **) &tmp1,sizeof(hipfftComplex)*totsz_pad);
    hipMalloc( (void **) &tmp2,sizeof(hipfftComplex)*totsz_pad);

    hipMalloc( (void **) &_sn,sizeof(float)*totsz);
    hipMalloc( (void **) &_r,sizeof(hipfftComplex)*numK*numsens);
	
    hipMalloc( (void **) &_ipk_we,sizeof(hipfftComplex)*numP*numK);
    hipMalloc( (void **) &_the_index,sizeof(int)*numP*numK);

    hipDeviceSynchronize();
   
    hipMemset( tmp1,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset( _img,0,sizeof(hipfftComplex)*totsz*numsens);
	 
     hipDeviceSynchronize();
 
  
     /************** copy data on device **********************/

	 mexPrintf("copying image data...\n");
     hipMemcpy( _img, img, sizeof(hipfftComplex)*numsens*totsz, hipMemcpyHostToDevice);
     mexPrintf("copying ipk we...\n");
	 hipMemcpy( _ipk_we, ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyHostToDevice);
     mexPrintf("copying the index...\n");
	 hipMemcpy( _the_index, the_index, sizeof(int)*numP*numK, hipMemcpyHostToDevice);
	 hipMemcpy( _sn, sn, sizeof(float)*totsz, hipMemcpyHostToDevice);
     
     hipMemcpy( ipk_we, _ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyDeviceToHost);
     hipMemcpy( the_index, _the_index, sizeof(int)*numP*numK, hipMemcpyDeviceToHost);
 
     hipDeviceSynchronize();
    
    if (VERBOSE == 1) 
        mexPrintf("numP: %i  numK: %i whd %i %i %i pad %i %i %i numsens: %i\n",numP,numK,w,h,d,w_pad,h_pad,d_pad,numsens);
            
      
    /************** copy bpidx on device **********************/
    int *_bpmidx;
    hipfftComplex *_bpweight;
    int *bpsize = (int*) malloc(sizeof(int)*numVox);
    int *bponset  = (int*) malloc(sizeof(int)*(numVox+1));
    int *_bpsize, *_bponset, *_bpidx;
    bponset[0] = 0;
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        bpsize[j] = mxGetM(Midx);
        bponset[j+1] = bponset[j] + bpsize[j];
    }
    
    int *tmp_bpmidx;
    hipfftComplex *tmp_bpweight;
    tmp_bpmidx = (int*) malloc(sizeof(int)*bponset[numVox]);
    tmp_bpweight = (hipfftComplex*) malloc(sizeof(hipfftComplex)*bponset[numVox]);
    if (tmp_bpmidx == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    if (tmp_bpweight == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        mxArray *Weight = mxGetCell(BPweight,j);
        int *midx = (int*)  mxGetData(Midx);
        hipfftComplex *bpwei = (hipfftComplex*) mxGetData(Weight);
        memcpy(tmp_bpmidx + bponset[j] , midx, sizeof(int)* bpsize[j]);
        memcpy(tmp_bpweight + bponset[j] , bpwei, sizeof(hipfftComplex)* bpsize[j]);    
    }
    
    hipMalloc( (void **) &_bpmidx,sizeof(int)* bponset[numVox]);
    hipMalloc( (void **) &_bpweight,sizeof(hipfftComplex)* bponset[numVox]);
    mexPrintf("copying tmp bpmidx...\n");  
    hipMemcpy(_bpmidx,tmp_bpmidx,sizeof(int)*bponset[numVox], hipMemcpyHostToDevice);
    mexPrintf("copying tmp bpweight...\n");
	hipMemcpy(_bpweight,tmp_bpweight,sizeof(hipfftComplex)*bponset[numVox], hipMemcpyHostToDevice);
 
    free(tmp_bpmidx);
    free(tmp_bpweight);

    hipMalloc( (void **) &_bpsize,sizeof(int)* numVox);   
    hipMalloc( (void **) &_bpidx,sizeof(int)* numVox);
    hipMalloc( (void **) &_bponset,sizeof(int)* numVox+1);    
    hipMemcpy(_bpsize,bpsize,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bpidx,bpidx,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bponset,bponset,sizeof(int)* numVox+1, hipMemcpyHostToDevice);
            
    GET_TIME(finish);

    
    if (VERBOSE == 1) {
        mexPrintf("num active Vox: %i\n",numVox);    
        mexPrintf("alloc/copy time: %f\n",finish-start);
    }
    
	int err;
	if (err=hipfftPlan3d(&plan, d_pad, h_pad, w_pad, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{
		mexPrintf("create cufft plan has failed with err %i \n",err);
		return;
	}
    // thread managements 
    int vx_block = 128;
    dim3 dimBlock_vx(vx_block,1);
    dim3 dimGrid_vx (numVox/vx_block + 1,1);
 
    dim3 dimBlock_dw(d,1);//d=64
    dim3 dimGrid_dw (w,h);//w=64,h=64

    dim3 dimBlock_sq(d,1);
    dim3 dimGrid_sq (w*h,1);
    
	// for sensing 
    int sens_block = 256;
    dim3 dimBlock_se(sens_block,1);
    dim3 dimGrid_se (numK/sens_block + 1,1);
   
     double AA_time = 0;
     double cg_time = 0;
     

    // we need this because first fft fails
    int _res = hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD);
	
    if (VERBOSE == 1)
      mexPrintf("first fft call ret: %i\n", _res);
	
	hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset(_r,0, sizeof(hipfftComplex)*numK*numsens);            
	
	mexPrintf("start forward gridding...\n");
    // do sens -- b=A x
    for (int i = 0; i < numsens; i++)
    { 
		//reset data for next coil
        hipMemset(tmp1,0, sizeof(hipfftComplex)*totsz_pad);
		
		//Multiplikation mit SN Matrix
        sn_mult<<<dimGrid_dw,dimBlock_dw>>>(tmp1,_img, _sn, w, h, d, w_pad, h_pad, d_pad);     
		
		//FT in k-space
        if (err=hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
        {
			mexPrintf("cufft has failed with err %i \n",err);
            return;
        }
		mexPrintf("hello, running...\n");
        hipMemset(_r,0, sizeof(hipfftComplex)*numK*numsens);
        dosens<<<dimGrid_se,dimBlock_se>>>(_r,tmp2,_ipk_we,_the_index,numP,numK);
            
		//add -> without sense 
		//addcoiltores<<<dimGrid_dw,dimBlock_dw>>>(_r,tmp2, totsz*numsens,i*totsz);
		//Quadratisch Summieren in Image Domain
		//quadradd<<<dimGrid_dw,dimBlock_dw>>>(_r,tmp2, w, h, d, w_pad, h_pad, d_pad);
     }
  
     hipMemcpy( res, _r, sizeof(hipfftComplex)*numK*numsens,hipMemcpyDeviceToHost);    
        
    if (VERBOSE == 1)
    {
        mexPrintf("\n");        
        mexPrintf(" AA time: %f \n",AA_time);
        mexPrintf(" cg  time: %f \n",cg_time);
    }

    hipFree(tmp1);
    hipFree(tmp2);
	
    hipFree(_r); 
    hipFree(_img);
	hipFree(_sn);
	
    hipFree(_ipk_we);
    hipFree(_the_index);
	
    hipFree(_bpmidx);
    hipFree(_bpweight);
    hipFree(_bpsize);
    hipFree(_bpidx);
    hipFree(_bponset);    
    
    hipfftDestroy(plan);
    free(bpsize);
    free(bponset);
 
     hipCtx_t  pctx ;
     hipCtxPopCurrent(&pctx);	
}













