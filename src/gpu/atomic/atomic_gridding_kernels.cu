#include "hip/hip_runtime.h"
#include "gridding_kernels.hpp"
#include "cuda_utils.cuh"

// convolve every data point on grid position -> controlled by threadIdx.x .y and .z 
// shared data holds grid values as software managed cache
//
// like non atomic gridding, but without temporaray grid
// atomic write to global output grid
// TODO evaluate performance
__global__ void convolutionKernelFromGrid(  DType* data, 
											DType* crds, 
											CufftType* gdata,
											DType* kernel, 
											int* sectors, 
											int* sector_centers
											)
{
	extern __shared__ DType sdata[];//externally managed shared memory

	int sec= blockIdx.x;
	//init shared memory
	for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
	{
			int y=threadIdx.y;
			int x=threadIdx.x;
			int s_ind = 2* getIndex(x,y,z,GI.sector_pad_width) ;
			sdata[s_ind] = 0.0f;//Re
			sdata[s_ind+1]=0.0f;//Im
	}
	__syncthreads();
	//start convolution
	if (sec < GI.sector_count)
	{
		int ind, k, i, j;
		__shared__ int max_x, max_y, max_z, imin, imax,jmin,jmax,kmin,kmax;

		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Positions controlled over thread inidices
		int data_cnt;
		data_cnt = sectors[sec];

		//loop over all data points of the current sector, and check if grid position lies inside 
		//affected region, if so, add data point weighted to grid position value
		while (data_cnt < sectors[sec+1])
		{
			__shared__ DType3 data_point; //datapoint shared in every thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];

			max_x = GI.sector_pad_width-1;
			max_y = GI.sector_pad_width-1;
			max_z = GI.sector_pad_width-1;

			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(&ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(&jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(&kz, &kmin, &kmax, max_z, GI.kernel_radius);
				                
			// grid this point onto its cartesian points neighbors
			for (k=threadIdx.z;k<=kmax; k += blockDim.z)
			{
				if (k<=kmax && k>=kmin)
				{
					kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
					dz_sqr = kz - data_point.z;
					dz_sqr *= dz_sqr;
					if (dz_sqr < GI.radiusSquared)
					{
						j=threadIdx.y;
						if (j<=jmax && j>=jmin)
						{
							jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
							dy_sqr = jy - data_point.y;
							dy_sqr *= dy_sqr;
							if (dy_sqr < GI.radiusSquared)	
							{
								i=threadIdx.x;
								
								if (i<=imax && i>=imin)
								{
									ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
									dx_sqr = ix - data_point.x;
									dx_sqr *= dx_sqr;
									if (dx_sqr < GI.radiusSquared)	
									{
										//get kernel value
										//Calculate Separable Filters 
										val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
											  kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
											  kernel[(int) round(dx_sqr * GI.dist_multiplier)];
										ind = 2* getIndex(i,j,k,GI.sector_pad_width);
								
										// multiply data by current kernel val 
										// grid complex or scalar 
										sdata[ind]   += val * data[2*data_cnt];
										sdata[ind+1] += val * data[2*data_cnt+1];
									} // kernel bounds check x, spherical support 
								} // x 	 
							} // kernel bounds check y, spherical support 
						} // y 
					} //kernel bounds check z 
				} // z
			}//for loop over z entries
			__syncthreads();
			data_cnt++;
		} //grid points per sector
	
		//write shared data to output grid
		__syncthreads();
		//int sector_ind_offset = sec * GI.sector_dim;
		int sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		//each thread position
		for (int z=threadIdx.z;z<GI.sector_pad_width; z += blockDim.z)
		{
			int y=threadIdx.y;
			int x=threadIdx.x;
			
			int s_ind = 2 * getIndex(x,y,z,GI.sector_pad_width) ;//index in shared grid
			int ind = sector_ind_offset + getIndex(x,y,z,GI.grid_width);//index in output grid
			
			if (isOutlier(x,y,z,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
				continue;

			atomicAdd(&(gdata[ind].x),sdata[s_ind]);//Re
			atomicAdd(&(gdata[ind].y),sdata[s_ind+1]);//Im
		}
	}//sec < sector_count	
}

__global__ void convolutionKernel2( DType* data, 
									DType* crds, 
									CufftType* gdata,
									DType* kernel, 
									int* sectors, 
									int* sector_centers
									)
{
	extern __shared__ DType sdata[];//externally managed shared memory
	int sec= blockIdx.x;

	if (sec < GI.sector_count)
	{
		//init shared memory
		for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind+= blockDim.x)
		{
			sdata[2*s_ind] = 0.0f;//Re
			sdata[2*s_ind+1]=0.0f;//Im
		}
		__syncthreads();
	
		//start convolution
		int ind, k, i, j;
		__shared__ int max_x, max_y, max_z, imin, imax,jmin,jmax,kmin,kmax;

		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		__shared__ int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Points over Threads
		int data_cnt = sectors[sec] + threadIdx.x;

		//loop over all data points of the current sector, and check if grid position lies inside 
		//affected region, if so, add data point weighted to grid position value
		while (data_cnt < sectors[sec+1])
		{
			__shared__ DType3 data_point; //datapoint shared in every thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];

			max_x = GI.sector_pad_width-1;
			max_y = GI.sector_pad_width-1;
			max_z = GI.sector_pad_width-1;

			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(&ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(&jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(&kz, &kmin, &kmax, max_z, GI.kernel_radius);
				                
			// grid this point onto its cartesian points neighbors
			k =kmin;
			while (k<=kmax && k>=kmin)
			{
				kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
				dz_sqr = kz - data_point.z;
				dz_sqr *= dz_sqr;
				if (dz_sqr < GI.radiusSquared)
				{
					j=jmin;
					while (j<=jmax && j>=jmin)
					{
						jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
						dy_sqr = jy - data_point.y;
						dy_sqr *= dy_sqr;
						if (dy_sqr < GI.radiusSquared)	
						{
							i= imin;						
							while (i<=imax && i>=imin)
							{
								ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
								dx_sqr = ix - data_point.x;
								dx_sqr *= dx_sqr;
								if (dx_sqr < GI.radiusSquared)	
								{
									//get kernel value
									//Calculate Separable Filters 
									val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
										  kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
										  kernel[(int) round(dx_sqr * GI.dist_multiplier)];
									ind = 2* getIndex(i,j,k,GI.sector_pad_width);
								
									// multiply data by current kernel val 
									// grid complex or scalar 
									atomicAdd(&(sdata[ind]),val * data[2*data_cnt]);
									atomicAdd(&(sdata[ind+1]),val * data[2*data_cnt+1]);
								} // kernel bounds check x, spherical support 
								i++;
							} // x 	 
						} // kernel bounds check y, spherical support 
						j++;
					} // y 
				} //kernel bounds check z 
				k++;
			} // z
			data_cnt = data_cnt + blockDim.x;
		} //grid points per sector
	
		//write shared data to output grid
		__syncthreads();
		//int sector_ind_offset = sec * GI.sector_dim;
		int sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		
		//each thread writes one position from shared mem to global mem
		for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind += blockDim.x)
		{
			/*if (sector_ind_offset + s_ind < GI.grid_width_dim)
			{
				atomicAdd(&(gdata[sector_ind_offset + s_ind].x),0.0f);
				atomicAdd(&(gdata[sector_ind_offset + s_ind+1].y),0.0f);
			}*/

			int z = (int)(s_ind / (GI.grid_width*GI.grid_width)) ;
			int r = s_ind - z * GI.grid_width * GI.grid_width;
			int y = (int)(r / GI.grid_width);
			int x = s_ind % GI.grid_width;

			int ind = sector_ind_offset + getIndex(x,y,z,GI.grid_width);//index in output grid
			
			if (isOutlier(x,y,z,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
				continue;

			atomicAdd(&(gdata[ind].x),1.0f);//sdata[2*s_ind]);//Re
			atomicAdd(&(gdata[ind].y),1.0f);//sdata[2*s_ind+1]);//Im
		}
	}//sec < sector_count	
}

//
// convolve every data point on grid position -> controlled by threadIdx.x .y and .z 
// shared data holds grid values as software managed cache
//
//
__global__ void convolutionKernel( DType* data, 
							    DType* crds, 
							    CufftType* gdata,
							    DType* kernel, 
							    int* sectors, 
								int* sector_centers
								)
{
	extern __shared__ DType sdata[]; //externally managed shared memory

	int  sec= blockIdx.x;
	//start convolution
	if (sec < GI.sector_count)
	{
		//shared???
		int ind, imin, imax, jmin, jmax,kmin,kmax, k, i, j;
		__shared__ int max_x;
		__shared__ int max_y;
		__shared__ int max_z;

		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Points over Threads
		int data_cnt = sectors[sec] + threadIdx.x;

		int sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		
		max_x = GI.sector_pad_width-1;
		max_y = GI.sector_pad_width-1;
		max_z = GI.sector_pad_width-1;
		
		while (data_cnt < sectors[sec+1])
		{
			DType3 data_point; //datapoint per thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];
			
			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(&ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(&jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(&kz, &kmin, &kmax, max_z, GI.kernel_radius);

			// convolve neighboring cartesian points to this data point
			k = kmin;
			while (k<=kmax && k>=kmin)
			{
				kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
				dz_sqr = kz - data_point.z;
				dz_sqr *= dz_sqr;
				
				if (dz_sqr < GI.radiusSquared)
				{
					j=jmin;
					while (j<=jmax && j>=jmin)
					{
						jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
						dy_sqr = jy - data_point.y;
						dy_sqr *= dy_sqr;
						if (dy_sqr < GI.radiusSquared)	
						{
							i=imin;								
							while (i<=imax && i>=imin)
							{
								ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
								dx_sqr = ix - data_point.x;
								dx_sqr *= dx_sqr;
								if (dx_sqr < GI.radiusSquared)	
								{
									// get kernel value
									//Berechnung mit Separable Filters 
									val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
											kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
											kernel[(int) round(dx_sqr * GI.dist_multiplier)];
									
									ind = sector_ind_offset + getIndex(i,j,k,GI.grid_width);//index in output grid
			
									if (isOutlier(i,j,k,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
									{
										i++;
										continue;
									}

									atomicAdd(&(gdata[ind].x),val * data[2*data_cnt]);//Re
									atomicAdd(&(gdata[ind].y),val * data[2*data_cnt+1]);//Im
								}// kernel bounds check x, spherical support 
								i++;
							} // x loop
						} // kernel bounds check y, spherical support  
						j++;
					} // y loop
				} //kernel bounds check z 
				k++;
			} // z loop
			data_cnt = data_cnt + blockDim.x;
		} //data points per sector
	} //sector check
}

void performConvolution( DType* data_d, 
						 DType* crds_d, 
						 CufftType* gdata_d,
						 DType* kernel_d, 
						 int* sectors_d, 
						 int* sector_centers_d,
						 DType* temp_gdata_d,
						 GriddingInfo* gi_host
						)
{
	//XXX other Kernel - slow //TODO evaluate

	/*long shared_mem_size = 2*gi_host->sector_dim*sizeof(DType);

	dim3 block_dim(gi_host->sector_pad_width,gi_host->sector_pad_width,N_THREADS_PER_SECTOR);
	dim3 grid_dim(gi_host->sector_count);
	
	printf("adjoint convolution requires %d bytes of shared memory!\n",shared_mem_size);
	convolutionKernelFromGrid<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d);
	*/
	//TODO how to calculate shared_mem_size???, shared_mem_needed?
	/*long shared_mem_size = 128 * sizeof(CufftType);//empiric

	dim3 block_dim(128);
	dim3 grid_dim(gi_host->sector_count);
	
	printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);
	convolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d);
	*/
	long shared_mem_size = 2*gi_host->sector_dim*sizeof(DType);

	dim3 block_dim(128);
	dim3 grid_dim(gi_host->sector_count);
	
	printf("adjoint convolution requires %d bytes of shared memory!\n",shared_mem_size);
	convolutionKernel2<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d);
	printf("...finished with: %s\n", hipGetErrorString(hipGetLastError()));
}

__global__ void forwardConvolutionKernel( CufftType* data, 
										  DType* crds, 
										  CufftType* gdata,
										  DType* kernel, 
										  int* sectors, 
										  int* sector_centers
											)
{
	extern __shared__ CufftType shared_out_data[];//externally managed shared memory
	//test
	CufftType out_data;


	int sec= blockIdx.x;
	//init shared memory
	//out_data[threadIdx.x].x = 0.0f;//Re
	//out_data[threadIdx.x].y = 0.0f;//Im

	out_data.x = 0.0f;//Re
	out_data.y = 0.0f;//Im

	__syncthreads();

	//start convolution
	if (sec < GI.sector_count)
	{
		//shared???
		int ind, max_x, max_y, max_z, imin, imax, jmin, jmax,kmin,kmax, k, i, j;
		DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

		int3 center;
		center.x = sector_centers[sec * 3];
		center.y = sector_centers[sec * 3 + 1];
		center.z = sector_centers[sec * 3 + 2];

		//Grid Points over Threads
		int data_cnt = sectors[sec] + threadIdx.x;
		//out_data[data_cnt].x = 0.0f;//Re
		//out_data[data_cnt].y = 0.0f;//Im
		out_data.x = 0.0f;//Re
		out_data.y = 0.0f;//Im
		//int sector_grid_offset = sec * GI.sector_dim;
		int sector_ind_offset = getIndex(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.grid_width);
		
		while (data_cnt < sectors[sec+1])
		{
			DType3 data_point; //datapoint per thread
			data_point.x = crds[3*data_cnt];
			data_point.y = crds[3*data_cnt +1];
			data_point.z = crds[3*data_cnt +2];

			max_x = GI.sector_pad_width-1;
			max_y = GI.sector_pad_width-1;
			max_z = GI.sector_pad_width-1;

			// set the boundaries of final dataset for gridding this point
			ix = (data_point.x + 0.5f) * (GI.grid_width) - center.x + GI.sector_offset;
			set_minmax(&ix, &imin, &imax, max_x, GI.kernel_radius);
			jy = (data_point.y + 0.5f) * (GI.grid_width) - center.y + GI.sector_offset;
			set_minmax(&jy, &jmin, &jmax, max_y, GI.kernel_radius);
			kz = (data_point.z + 0.5f) * (GI.grid_width) - center.z + GI.sector_offset;
			set_minmax(&kz, &kmin, &kmax, max_z, GI.kernel_radius);

			// convolve neighboring cartesian points to this data point
			k = kmin;
			//data[data_cnt].x = -1.0f;//(float)k;
			//data[data_cnt].y =-1.0f;// (float)kmax;
			while (k<=kmax && k>=kmin)
			{
				kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;//(k - center_z) *width_inv;
				dz_sqr = kz - data_point.z;
				dz_sqr *= dz_sqr;
				
				//data[data_cnt].x = kz;
				//data[data_cnt].y = center.z;//static_cast<DType>((k + center.z - 8 - GI.sector_offset)) / static_cast<DType>((GI.grid_width)) - 0.5f;
				if (dz_sqr < GI.radiusSquared)
				{
					//data[data_cnt].x = 1.1f;
					//data[data_cnt].y = 1.1f;
					j=jmin;
					while (j<=jmax && j>=jmin)
					{
						//data[data_cnt].x = 1.2f;
						//data[data_cnt].y = 1.2f;
						jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;   //(j - center_y) *width_inv;
						dy_sqr = jy - data_point.y;
						dy_sqr *= dy_sqr;
						if (dy_sqr < GI.radiusSquared)	
						{
							//data[data_cnt].x = 1.3f;
							//data[data_cnt].y = 1.3f;
							i=imin;								
							while (i<=imax && i>=imin)
							{
								//data[data_cnt].x = 1.4f;
								//data[data_cnt].y = 1.4f;
								ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.grid_width)) - 0.5f;// (i - center_x) *width_inv;
								dx_sqr = ix - data_point.x;
								dx_sqr *= dx_sqr;
								if (dx_sqr < GI.radiusSquared)	
								{
									//data[data_cnt].x = 1.5f;
									//data[data_cnt].y = 1.5f;
									// get kernel value
									//Berechnung mit Separable Filters 
									val = kernel[(int) round(dz_sqr * GI.dist_multiplier)] *
											kernel[(int) round(dy_sqr * GI.dist_multiplier)] *
											kernel[(int) round(dx_sqr * GI.dist_multiplier)];
									
									ind = (sector_ind_offset + getIndex(i,j,k,GI.grid_width));

									// multiply data by current kernel val 
									// grid complex or scalar 
									if (isOutlier(i,j,k,center.x,center.y,center.z,GI.grid_width,GI.sector_offset))
									{
										i++;
										continue;
									}
				
									//out_data[data_cnt].x = 1.0f; //val * gdata[ind].x;
									//out_data[data_cnt].y = 1.0f; //val * gdata[ind].y;		
									out_data.x += gdata[ind].x * val; //+= /*val **/ gdata[ind].x;
									out_data.y += gdata[ind].y * val; //+= /*val **/ gdata[ind].y;
									
								}// kernel bounds check x, spherical support 
								i++;
							} // x loop
						} // kernel bounds check y, spherical support  
						j++;
					} // y loop
				} //kernel bounds check z 
				k++;
			} // z loop
			//data[data_cnt] = out_data[data_cnt];
			data[data_cnt].x = out_data.x;// / sqrt((DType)GI.kernel_width*GI.kernel_width*GI.kernel_width);
			data[data_cnt].y = out_data.y;// / sqrt((DType)GI.kernel_width*GI.kernel_width*GI.kernel_width);
			
			data_cnt = data_cnt + blockDim.x;

			//out_data[data_cnt].x = (DType)0.0f;
			//out_data[data_cnt].y = (DType)0.0f;
			out_data.x = 0.0f;//Re
			out_data.y = 0.0f;//Im
			//data[data_cnt] = out_data[data_cnt];
			//data_cnt++;
		} //data points per sector
	} //sector check
}

void performForwardConvolution( CufftType*		data_d, 
								DType*			crds_d, 
								CufftType*		gdata_d,
								DType*			kernel_d, 
								int*			sectors_d, 
								int*			sector_centers_d,
								GriddingInfo*	gi_host
								)
{
	//TODO how to calculate shared_mem_size???, shared_mem_needed?
	long shared_mem_size = 128 * sizeof(CufftType);//empiric

	dim3 block_dim(128);
	dim3 grid_dim(gi_host->sector_count);
	
	printf("convolution requires %d bytes of shared memory!\n",shared_mem_size);
	forwardConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,kernel_d,sectors_d,sector_centers_d);
}
