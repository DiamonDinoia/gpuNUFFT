#include "hip/hip_runtime.h"
#ifndef TEXTURE_GPUNUFFT_KERNELS_H
#define TEXTURE_GPUNUFFT_KERNELS_H
#include "gpuNUFFT_kernels.hpp"
#include "../std_gpuNUFFT_kernels.cu"
#include "cuda_utils.cuh"

// ----------------------------------------------------------------------------
// convolutionKernel: NUFFT^H kernel
//
// Performs the gpuNUFFT step by convolution of sample points with 
// interpolation function and resampling onto grid. Basic concept based on Zwart
// et al. 
//
// parameters:
//  * data           : complex input sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : output grid data 
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * temp_gdata     : temporary grid data
//  * N              : number of threads
__device__ void textureConvolutionFunction(int* sec, int sec_max, int sec_offset, DType2* sdata, DType2* data, DType* crds, CufftType* gdata, IndType* sectors, IndType* sector_centers)
{
  //start convolution
  int ind, k, i, j, x, y, z;
  int imin, imax,jmin,jmax,kmin,kmax;

  DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

  __shared__ IndType3 center;
  center.x = sector_centers[sec[threadIdx.x] * 3];
  center.y = sector_centers[sec[threadIdx.x] * 3 + 1];
  center.z = sector_centers[sec[threadIdx.x] * 3 + 2];

  //Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
  //loop over all data points of the current sector, and check if grid position lies inside 
  //affected region, if so, add data point weighted to grid position value
  while (data_cnt < sec_max)
  {
    DType3 data_point; //datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt +GI.data_count];
    data_point.z = crds[data_cnt +2*GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = (data_point.y + 0.5f) * (GI.gridDims.y) - center.y + GI.sector_offset;
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);
    kz = (data_point.z + 0.5f) * (GI.gridDims.z) - center.z + GI.sector_offset;
    set_minmax(&kz, &kmin, &kmax, GI.sector_pad_max, GI.kernel_radius);

    // grid this point onto its cartesian points neighbors
    k =kmin;
    while (k<=kmax && k>=kmin)
    {
      kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.z)) - 0.5f;//(k - center_z) *width_inv;
      dz_sqr = (kz - data_point.z)*GI.aniso_z_scale;
      dz_sqr *= dz_sqr;
      j=jmin;
      while (j<=jmax && j>=jmin)
      {
        jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;   //(j - center_y) *width_inv;
        dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
        dy_sqr *= dy_sqr;
        i= imin;						
        while (i<=imax && i>=imin)
        {
          ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;// (i - center_x) *width_inv;
          dx_sqr = (ix - data_point.x)*GI.aniso_x_scale;
          dx_sqr *= dx_sqr;
          //get kernel value
          val = computeTextureLookup(dx_sqr*GI.radiusSquared_inv,dy_sqr*GI.radiusSquared_inv,dz_sqr*GI.radiusSquared_inv);

          ind = getIndex(i,j,k,GI.sector_pad_width);

          // multiply data by current kernel val 
          // grid complex or scalar 
          atomicAdd(&(sdata[ind].x),val * tex1Dfetch(texDATA,data_cnt).x);
          atomicAdd(&(sdata[ind].y),val * tex1Dfetch(texDATA,data_cnt).y);
          i++;
        } // x 	 
        j++;
      } // y 
      k++;
    } // z
    data_cnt = data_cnt + blockDim.x;
  } //grid points per sector

  //write shared data to output grid
  __syncthreads();
  //int sector_ind_offset = sec * GI.sector_dim;
  __shared__ int sector_ind_offset;
  sector_ind_offset  = computeXYZ2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.gridDims);

  //each thread writes one position from shared mem to global mem
  for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind += blockDim.x)
  {
    getCoordsFromIndex(s_ind,&x,&y,&z,GI.sector_pad_width);

    if (isOutlier(x,y,z,center.x,center.y,center.z,GI.gridDims,GI.sector_offset))
      //calculate opposite index
      ind = computeXYZ2Lin(calculateOppositeIndex(x,center.x,GI.gridDims.x,GI.sector_offset),
      calculateOppositeIndex(y,center.y,GI.gridDims.y,GI.sector_offset),
      calculateOppositeIndex(z,center.z,GI.gridDims.z,GI.sector_offset),
      GI.gridDims);
    else
      ind = sector_ind_offset + computeXYZ2Lin(x,y,z,GI.gridDims);//index in output grid

    atomicAdd(&(gdata[ind].x),sdata[s_ind].x);//Re
    atomicAdd(&(gdata[ind].y),sdata[s_ind].y);//Im

    //reset shared mem
    sdata[s_ind].x = (DType)0.0;
    sdata[s_ind].y = (DType)0.0;
  }
}

__global__ void textureConvolutionKernel(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory
  
  //init shared memory
  for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind+= blockDim.x)
  {
    sdata[s_ind].x = (DType)0.0;//Re
    sdata[s_ind].y = (DType)0.0;//Im
  }
  __syncthreads();

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x] = blockIdx.x;
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x]+1];
    textureConvolutionFunction(sec,data_max,0,sdata,data,crds,gdata,sectors,sector_centers);
    __syncthreads();
    sec[threadIdx.x] = sec[threadIdx.x]+ gridDim.x;
  }//sec < sector_count	
}

__global__ void balancedTextureConvolutionKernel(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType2* sector_processing_order,
  IndType* sector_centers,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory

  //init shared memory
  for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind+= blockDim.x)
  {
    sdata[s_ind].x = (DType)0.0;//Re
    sdata[s_ind].y = (DType)0.0;//Im
  }
  __syncthreads();

  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];
  
  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x]+1],sectors[sec[threadIdx.x]] + threadIdx.x + sector_processing_order[sec_cnt].y+MAXIMUM_PAYLOAD);
    textureConvolutionFunction(sec,data_max,sector_processing_order[sec_cnt].y,sdata,data,crds,gdata,sectors,sector_centers);
    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  }//sec < sector_count	
}


// ----------------------------------------------------------------------------
// convolutionKernel: NUFFT^H kernel
//
// Performs the gpuNUFFT step by convolution of sample points with 
// interpolation function and resampling onto grid. Basic concept based on Zwart
// et al. 
//
// parameters:
//  * data           : complex input sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : output grid data 
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * temp_gdata     : temporary grid data
//  * N              : number of threads
__device__ void textureConvolutionFunction2D(DType2* sdata,int* sec, int sec_max, int sec_offset, DType2* data, DType* crds, CufftType* gdata,IndType* sectors, IndType* sector_centers)
{
  //start convolution
  int ind, i, j, x, y;
  int imin, imax,jmin,jmax;

  DType dx_sqr, dy_sqr, val, ix, jy;

  __shared__ IndType2 center;
  center.x = sector_centers[sec[threadIdx.x] * 2];
  center.y = sector_centers[sec[threadIdx.x] * 2 + 1];

  //Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
  //loop over all data points of the current sector, and check if grid position lies inside 
  //affected region, if so, add data point weighted to grid position value
  while (data_cnt < sec_max)
  {
    DType2 data_point; //datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt +GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = (data_point.y + 0.5f) * (GI.gridDims.y) - center.y + GI.sector_offset;
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

    // grid this point onto its cartesian points neighbors
    j=jmin;
    while (j<=jmax && j>=jmin)
    {
      jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;   //(j - center_y) *width_inv;
      dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
      dy_sqr *= dy_sqr;
      i= imin;						
      while (i<=imax && i>=imin)
      {
        ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;// (i - center_x) *width_inv;
        dx_sqr = (ix - data_point.x)*GI.aniso_x_scale;
        dx_sqr *= dx_sqr;
        //get kernel value
        //Calculate Separable Filters 
        val = computeTextureLookup(dx_sqr*GI.radiusSquared_inv,dy_sqr*GI.radiusSquared_inv);

        ind = getIndex2D(i,j,GI.sector_pad_width);

        // multiply data by current kernel val 
        // grid complex or scalar
        atomicAdd(&(sdata[ind].x),val * tex1Dfetch(texDATA,data_cnt).x);
        atomicAdd(&(sdata[ind].y),val * tex1Dfetch(texDATA,data_cnt).y);
        i++;
      } // x 	 
      j++;
    } // y 
    data_cnt = data_cnt + blockDim.x;
  } //grid points per sector

  //write shared data to output grid
  __syncthreads();
  //int sector_ind_offset = sec * GI.sector_dim;
  __shared__ int sector_ind_offset;
  sector_ind_offset  = computeXY2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,GI.gridDims);

  //each thread writes one position from shared mem to global mem
  for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind += blockDim.x)
  {
    getCoordsFromIndex2D(s_ind,&x,&y,GI.sector_pad_width);

    if (isOutlier2D(x,y,center.x,center.y,GI.gridDims,GI.sector_offset))
      //calculate opposite index
      ind = computeXY2Lin(calculateOppositeIndex(x,center.x,GI.gridDims.x,GI.sector_offset),
      calculateOppositeIndex(y,center.y,GI.gridDims.y,GI.sector_offset),
      GI.gridDims);
    else
      ind = sector_ind_offset + computeXY2Lin(x,y,GI.gridDims);//index in output grid

    atomicAdd(&(gdata[ind].x),sdata[s_ind].x);//Re
    atomicAdd(&(gdata[ind].y),sdata[s_ind].y);//Im
    
    //reset shared mem
    sdata[s_ind].x = (DType)0.0;
    sdata[s_ind].y = (DType)0.0;
  }
}
__global__ void textureConvolutionKernel2D(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory
  
  //init shared memory
  for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind+= blockDim.x)
  {
    sdata[s_ind].x = (DType)0.0;//Re
    sdata[s_ind].y = (DType)0.0;//Im
  }
  __syncthreads();

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x] = blockIdx.x;
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x]+1];
    textureConvolutionFunction2D(sdata,sec,data_max,0,data,crds,gdata,sectors,sector_centers);
    __syncthreads();
    sec[threadIdx.x] = sec[threadIdx.x]+ gridDim.x;
  }//sec < sector_count	
}

__global__ void balancedTextureConvolutionKernel2D(DType2* data, 
  DType* crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType2* sector_processing_order,
  IndType* sector_centers,
  int N
  )
{
  extern __shared__ DType2 sdata[];//externally managed shared memory
  
  //init shared memory
  for (int s_ind=threadIdx.x;s_ind<GI.sector_dim; s_ind+= blockDim.x)
  {
    sdata[s_ind].x = (DType)0.0;//Re
    sdata[s_ind].y = (DType)0.0;//Im
  }
  __syncthreads();

  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];
  
  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x; 
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x]+1],sectors[sec[threadIdx.x]] + threadIdx.x + sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);
    textureConvolutionFunction2D(sdata,sec,data_max,sector_processing_order[sec_cnt].y,data,crds,gdata,sectors,sector_centers);
    __syncthreads();
    sec_cnt = sec_cnt+ gridDim.x;
  }//sec < sector_count	
}

void performTextureConvolution( DType2* data_d, 
  DType* crds_d, 
  CufftType* gdata_d,
  DType*			kernel_d, 
  IndType* sectors_d, 
  IndType* sector_centers_d,
  gpuNUFFT::GpuNUFFTInfo* gi_host
  )
{
  long shared_mem_size = (gi_host->sector_dim)*sizeof(DType2);
  int thread_size =THREAD_BLOCK_SIZE;

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,1));
  if (DEBUG)
  {
    printf("adjoint texture convolution requires %d bytes of shared memory!\n",shared_mem_size);
    printf("grid dim %d, block dim %d \n",grid_dim.x, block_dim.x); 
  }
  if (gi_host->is2Dprocessing)
    textureConvolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,gi_host->sector_count);
  else
    textureConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,gi_host->sector_count);

  if (DEBUG)
    printf("...finished with: %s\n", hipGetErrorString(hipGetLastError()));
}

void performTextureConvolution( DType2* data_d, 
  DType* crds_d, 
  CufftType* gdata_d,
  DType*			kernel_d, 
  IndType* sectors_d, 
  IndType2* sector_processing_order_d,
  IndType* sector_centers_d,
  gpuNUFFT::GpuNUFFTInfo* gi_host
  )
{
  long shared_mem_size = (gi_host->sector_dim)*sizeof(DType2);
  int thread_size =THREAD_BLOCK_SIZE;

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,1));
  if (DEBUG)
  {
    printf("adjoint balanced texture convolution requires %d bytes of shared memory!\n",shared_mem_size);
    printf("grid dim %d, block dim %d \n",grid_dim.x, block_dim.x); 
  }
  if (gi_host->is2Dprocessing)
    balancedTextureConvolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_processing_order_d,sector_centers_d,gi_host->sectorsToProcess);
  else
    balancedTextureConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_processing_order_d,sector_centers_d,gi_host->sectorsToProcess);

  if (DEBUG)
    printf("...finished with: %s\n", hipGetErrorString(hipGetLastError()));
}

// ----------------------------------------------------------------------------
// forwardConvolutionKernel: NUFFT kernel
//
// Performs the inverse gpuNUFFT step by convolution of grid points with 
// interpolation function and resampling onto trajectory. 
//
// parameters:
//  * data           : complex output sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : input grid data 
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * N              : number of threads

__device__ void textureForwardConvolutionFunction(int* sec, int sec_max, int sec_offset, DType2* sdata, CufftType* gdata_cache, DType2* data, DType* crds, CufftType* gdata, IndType* sectors, IndType* sector_centers)
{
  int ind, imin, imax, jmin, jmax,kmin,kmax, k, i, j;
  DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

  __shared__ IndType3 center;
  center.x = sector_centers[sec[threadIdx.x] * 3];
  center.y = sector_centers[sec[threadIdx.x] * 3 + 1];
  center.z = sector_centers[sec[threadIdx.x] * 3 + 2];

  __shared__ int sector_ind_offset;
  sector_ind_offset = computeXYZ2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,center.z - GI.sector_offset,GI.gridDims);

  // init sector cache 
  // preload sector grid data into cache
  for (int ind=threadIdx.x; ind<GI.sector_dim; ind+=blockDim.x)
  {
    int grid_index;
    getCoordsFromIndex(ind,&i,&j,&k,GI.sector_pad_width);

    if (isOutlier(i,j,k,center.x,center.y,center.z,GI.gridDims,GI.sector_offset))
      //calculate opposite index
      grid_index = computeXYZ2Lin(calculateOppositeIndex(i,center.x,GI.gridDims.x,GI.sector_offset),
      calculateOppositeIndex(j,center.y,GI.gridDims.y,GI.sector_offset),
      calculateOppositeIndex(k,center.z,GI.gridDims.z,GI.sector_offset),
      GI.gridDims);
    else
      grid_index = (sector_ind_offset + computeXYZ2Lin(i,j,k,GI.gridDims));

    gdata_cache[ind].x = tex1Dfetch(texGDATA,grid_index).x;
    gdata_cache[ind].y = tex1Dfetch(texGDATA,grid_index).y;
  }
    
  __syncthreads();

  //Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
   
  while (data_cnt < sec_max)
  {
    DType3 data_point; //datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];
    data_point.z = crds[data_cnt + 2*GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = (data_point.y + 0.5f) * (GI.gridDims.y) - center.y + GI.sector_offset;
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);
    kz = (data_point.z + 0.5f) * (GI.gridDims.z) - center.z + GI.sector_offset;
    set_minmax(&kz, &kmin, &kmax, GI.sector_pad_max, GI.kernel_radius);

    // convolve neighboring cartesian points to this data point
    k = kmin;
    while (k<=kmax && k>=kmin)
    {
      kz = static_cast<DType>((k + center.z - GI.sector_offset)) / static_cast<DType>((GI.gridDims.z)) - 0.5f;
      dz_sqr = (kz - data_point.z)*GI.aniso_z_scale;
      dz_sqr *= dz_sqr;

      j=jmin;
      while (j<=jmax && j>=jmin)
      {
        jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;
        dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
        dy_sqr *= dy_sqr;
        i=imin;								
        while (i<=imax && i>=imin)
        {
          ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;
          dx_sqr = (ix - data_point.x)*GI.aniso_x_scale;
          dx_sqr *= dx_sqr;
          
          // get kernel value
          val = computeTextureLookup(dx_sqr*GI.radiusSquared_inv,dy_sqr*GI.radiusSquared_inv,dz_sqr*GI.radiusSquared_inv);

          ind = getIndex(i,j,k,GI.sector_pad_width);

          sdata[threadIdx.x].x += gdata_cache[ind].x * val; 
          sdata[threadIdx.x].y += gdata_cache[ind].y * val;									

          i++;
        } // x loop
        j++;
      } // y loop
      k++;
    } // z loop
    atomicAdd(&(data[data_cnt].x),sdata[threadIdx.x].x);
    atomicAdd(&(data[data_cnt].y),sdata[threadIdx.x].y);

    data_cnt = data_cnt + blockDim.x;

    sdata[threadIdx.x].x = (DType)0.0;//Re
    sdata[threadIdx.x].y = (DType)0.0;//Im
  } //data points per sector
}

__global__ void textureForwardConvolutionKernel(CufftType* data, 
  DType*     crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  int N)
{
  extern __shared__ CufftType shared[];//externally managed shared memory
  CufftType* shared_out_data =(CufftType*) &shared[0];  
  CufftType* gdata_cache =(CufftType*) &shared[blockDim.x]; 

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x]= blockIdx.x;
  
  //init shared memory
  shared_out_data[threadIdx.x].x = (DType)0.0;//Re
  shared_out_data[threadIdx.x].y = (DType)0.0;//Im

  __syncthreads();
  //start convolution
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x]+1];	

    textureForwardConvolutionFunction(sec,data_max,0,shared_out_data,gdata_cache,data,crds,gdata,sectors,sector_centers);
    __syncthreads();
    sec[threadIdx.x]= sec[threadIdx.x] + gridDim.x;
  } //sector check
}

__global__ void balancedTextureForwardConvolutionKernel(CufftType* data, 
  DType*     crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType2* sector_processing_order,
  IndType* sector_centers,
  int N)
{
  extern __shared__ CufftType shared[];//externally managed shared memory
  CufftType* shared_out_data =(CufftType*) &shared[0];  
  CufftType* gdata_cache =(CufftType*) &shared[blockDim.x]; 
  
  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];

  //init shared memory
  shared_out_data[threadIdx.x].x = (DType)0.0;//Re
  shared_out_data[threadIdx.x].y = (DType)0.0;//Im

  __syncthreads();
  //start convolution
  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x]+1],sectors[sec[threadIdx.x]] + threadIdx.x + sector_processing_order[sec_cnt].y+MAXIMUM_PAYLOAD);
       
    textureForwardConvolutionFunction(sec,data_max,sector_processing_order[sec_cnt].y,shared_out_data,gdata_cache,data,crds,gdata,sectors,sector_centers);
    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  } //sector check
}

__device__ void textureForwardConvolutionFunction2D(int* sec, int sec_max, int sec_offset, DType2* sdata, CufftType* gdata_cache, DType2* data, DType* crds, CufftType* gdata, IndType* sectors, IndType* sector_centers)
{
  int ind, imin, imax, jmin, jmax, i, j;
  DType dx_sqr, dy_sqr, val, ix, jy;

  __shared__ IndType2 center;
  center.x = sector_centers[sec[threadIdx.x] * 2];
  center.y = sector_centers[sec[threadIdx.x] * 2 + 1];

  __shared__ int sector_ind_offset;
  sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,center.y - GI.sector_offset,GI.gridDims);

    // init sector cache 
  // preload sector grid data into cache
  for (int ind=threadIdx.x; ind<GI.sector_dim; ind+=blockDim.x)
  {
    int grid_index;
    getCoordsFromIndex2D(ind,&i,&j,GI.sector_pad_width);

    // multiply data by current kernel val 
    // grid complex or scalar 
    if (isOutlier2D(i,j,center.x,center.y,GI.gridDims.x,GI.sector_offset))
      //calculate opposite index
      grid_index = getIndex2D(calculateOppositeIndex(i,center.x,GI.gridDims.x,GI.sector_offset),
      calculateOppositeIndex(j,center.y,GI.gridDims.y,GI.sector_offset),
      GI.gridDims.x);
    else
      grid_index = (sector_ind_offset + getIndex2D(i,j,GI.gridDims.x));

    gdata_cache[ind].x = tex1Dfetch(texGDATA,grid_index).x;
    gdata_cache[ind].y = tex1Dfetch(texGDATA,grid_index).y;
  }
  __syncthreads();

  //Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
    
  while (data_cnt < sec_max)
  {
    DType2 data_point; //datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = (data_point.x + 0.5f) * (GI.gridDims.x) - center.x + GI.sector_offset;
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = (data_point.y + 0.5f) * (GI.gridDims.y) - center.y + GI.sector_offset;
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

    // convolve neighboring cartesian points to this data point
    j=jmin;
    while (j<=jmax && j>=jmin)
    {
      jy = static_cast<DType>(j + center.y - GI.sector_offset) / static_cast<DType>((GI.gridDims.y)) - 0.5f;   //(j - center_y) *width_inv;
      dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
      dy_sqr *= dy_sqr;
      i=imin;								
      while (i<=imax && i>=imin)
      {
        ix = static_cast<DType>(i + center.x - GI.sector_offset) / static_cast<DType>((GI.gridDims.x)) - 0.5f;// (i - center_x) *width_inv;
        dx_sqr = (ix - data_point.x)*GI.aniso_x_scale;
        dx_sqr *= dx_sqr;
        // get kernel value
        // calc as separable filter
        val = computeTextureLookup(dx_sqr*GI.radiusSquared_inv,dy_sqr*GI.radiusSquared_inv);
   
        ind = getIndex2D(i,j,GI.sector_pad_width);

        sdata[threadIdx.x].x += gdata_cache[ind].x * val; 
        sdata[threadIdx.x].y += gdata_cache[ind].y * val;									
        i++;
      } // x loop
    j++;
    } // y loop
    atomicAdd(&(data[data_cnt].x),sdata[threadIdx.x].x);
    atomicAdd(&(data[data_cnt].y),sdata[threadIdx.x].y);

    data_cnt = data_cnt + blockDim.x;

    sdata[threadIdx.x].x = (DType)0.0;//Re
    sdata[threadIdx.x].y = (DType)0.0;//Im
  } //data points per sector
}

__global__ void textureForwardConvolutionKernel2D(CufftType* data, 
  DType*     crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType* sector_centers,
  int N)
{
  extern __shared__ CufftType shared[];//externally managed shared memory
  CufftType* shared_out_data =(CufftType*) &shared[0];  
  CufftType* gdata_cache =(CufftType*) &shared[blockDim.x]; 

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x]= blockIdx.x;

  //init shared memory
  shared_out_data[threadIdx.x].x = (DType)0.0;//Re
  shared_out_data[threadIdx.x].y = (DType)0.0;//Im
  __syncthreads();
  //start convolution
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x]+1];

    textureForwardConvolutionFunction2D(sec,data_max,0,shared_out_data,gdata_cache,data,crds,gdata,sectors,sector_centers);

    __syncthreads();
    sec[threadIdx.x]= sec[threadIdx.x] + gridDim.x;
  } //sector check
}

__global__ void balancedTextureForwardConvolutionKernel2D(CufftType* data, 
  DType*     crds, 
  CufftType* gdata,
  IndType* sectors, 
  IndType2* sector_processing_order,
  IndType* sector_centers,
  int N)
{
  extern __shared__ CufftType shared[];//externally managed shared memory
  CufftType* shared_out_data =(CufftType*) &shared[0];  
  CufftType* gdata_cache =(CufftType*) &shared[blockDim.x]; 

  int sec_cnt= blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];
  
  //init shared memory
  shared_out_data[threadIdx.x].x = (DType)0.0;//Re
  shared_out_data[threadIdx.x].y = (DType)0.0;//Im
  __syncthreads();
  //start convolution
  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x]+1],sectors[sec[threadIdx.x]] + threadIdx.x + sector_processing_order[sec_cnt].y+MAXIMUM_PAYLOAD);
    
    textureForwardConvolutionFunction2D(sec,data_max,sector_processing_order[sec_cnt].y,shared_out_data,gdata_cache,data,crds,gdata,sectors,sector_centers);

    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  } //sector check
}


void performTextureForwardConvolution( CufftType*		data_d, 
  DType*			crds_d, 
  CufftType*		gdata_d,
  DType*			kernel_d, 
  IndType*		sectors_d, 
  IndType*		sector_centers_d,
  gpuNUFFT::GpuNUFFTInfo*	gi_host
  )
{
  int thread_size =THREAD_BLOCK_SIZE;
  long shared_mem_size = (thread_size + gi_host->sector_dim) * sizeof(CufftType);//empiric

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,thread_size));

  if (DEBUG)
    printf("texture forward convolution requires %d bytes of shared memory!\n",shared_mem_size);
  if (gi_host->is2Dprocessing)
    textureForwardConvolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,gi_host->sector_count);
  else
    textureForwardConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_centers_d,gi_host->sector_count);
}

void performTextureForwardConvolution( CufftType*		data_d, 
  DType*			crds_d, 
  CufftType*  gdata_d,
  DType*			kernel_d, 
  IndType*		sectors_d, 
  IndType2*   sector_processing_order_d,
  IndType*		sector_centers_d,
  gpuNUFFT::GpuNUFFTInfo*	gi_host
  )
{
  int thread_size =THREAD_BLOCK_SIZE;
  long shared_mem_size = (thread_size + gi_host->sector_dim) * sizeof(CufftType);//empiric

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count,thread_size));

  if (DEBUG)
    printf("balanced texture forward convolution requires %d bytes of shared memory!\n",shared_mem_size);
  if (gi_host->is2Dprocessing)
    balancedTextureForwardConvolutionKernel2D<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_processing_order_d,sector_centers_d,gi_host->sectorsToProcess);
  else
    balancedTextureForwardConvolutionKernel<<<grid_dim,block_dim,shared_mem_size>>>(data_d,crds_d,gdata_d,sectors_d,sector_processing_order_d,sector_centers_d,gi_host->sectorsToProcess);
}

#endif
